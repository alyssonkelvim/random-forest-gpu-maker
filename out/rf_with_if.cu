#include "hip/hip_runtime.h"
__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, float *F6, float *F7, float *F8, float *F9, float *F10, int *P, const int N)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[6]; 
	Class[0] = 0;
	Class[1] = 0;
	Class[2] = 0;
	Class[3] = 0;
	Class[4] = 0;
	Class[5] = 0;	if (i < N) {
		if (F7[i] <= 0.995) {
			if (F1[i] <= 0.335) {
				if (F1[i] <= 0.275) {
					if (F0[i] <= 6.0) {
						Class[2]++;
					} else {
						if (F6[i] <= 16.0) {
							if (F8[i] <= 3.215) {
								Class[3]++;
							} else {
								Class[4]++;
							}
						} else {
							if (F4[i] <= 0.076) {
								Class[3]++;
							} else {
								if (F8[i] <= 3.12) {
									Class[5]++;
								} else {
									Class[3]++;
								}
							}
						}
					}
				} else {
					if (F7[i] <= 0.995) {
						if (F0[i] <= 8.0) {
							if (F0[i] <= 7.75) {
								Class[4]++;
							} else {
								if (F8[i] <= 3.15) {
									Class[4]++;
								} else {
									Class[3]++;
								}
							}
						} else {
							Class[4]++;
						}
					} else {
						if (F0[i] <= 8.8) {
							Class[2]++;
						} else {
							Class[4]++;
						}
					}
				}
			} else {
				if (F1[i] <= 1.015) {
					if (F6[i] <= 88.5) {
						if (F9[i] <= 0.525) {
							if (F1[i] <= 0.83) {
								if (F2[i] <= 0.285) {
									if (F0[i] <= 7.2) {
										Class[2]++;
									} else {
										if (F1[i] <= 0.625) {
											Class[2]++;
										} else {
											Class[4]++;
										}
									}
								} else {
									Class[4]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F8[i] <= 3.27) {
								if (F0[i] <= 7.6) {
									if (F7[i] <= 0.995) {
										if (F3[i] <= 1.9) {
											Class[5]++;
										} else {
											if (F2[i] <= 0.315) {
												Class[5]++;
											} else {
												Class[4]++;
											}
										}
									} else {
										Class[2]++;
									}
								} else {
									if (F10[i] <= 11.3) {
										Class[3]++;
									} else {
										if (F0[i] <= 8.85) {
											Class[5]++;
										} else {
											Class[4]++;
										}
									}
								}
							} else {
								if (F8[i] <= 3.515) {
									if (F10[i] <= 12.65) {
										if (F8[i] <= 3.415) {
											Class[3]++;
										} else {
											if (F2[i] <= 0.155) {
												if (F3[i] <= 2.35) {
													Class[3]++;
												} else {
													if (F6[i] <= 40.5) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												}
											} else {
												if (F5[i] <= 9.5) {
													Class[1]++;
												} else {
													Class[4]++;
												}
											}
										}
									} else {
										if (F4[i] <= 0.066) {
											Class[3]++;
										} else {
											Class[4]++;
										}
									}
								} else {
									if (F5[i] <= 13.5) {
										if (F7[i] <= 0.995) {
											if (F6[i] <= 22.5) {
												Class[2]++;
											} else {
												if (F2[i] <= 0.03) {
													Class[4]++;
												} else {
													Class[5]++;
												}
											}
										} else {
											Class[3]++;
										}
									} else {
										Class[3]++;
									}
								}
							}
						}
					} else {
						if (F3[i] <= 2.25) {
							if (F1[i] <= 0.525) {
								Class[4]++;
							} else {
								if (F1[i] <= 0.688) {
									Class[3]++;
								} else {
									Class[4]++;
								}
							}
						} else {
							if (F0[i] <= 6.15) {
								Class[2]++;
							} else {
								Class[3]++;
							}
						}
					}
				} else {
					if (F8[i] <= 3.665) {
						if (F9[i] <= 0.415) {
							Class[0]++;
						} else {
							Class[1]++;
						}
					} else {
						Class[2]++;
					}
				}
			}
		} else {
			if (F10[i] <= 9.85) {
				if (F6[i] <= 83.5) {
					if (F9[i] <= 0.575) {
						if (F6[i] <= 80.0) {
							if (F7[i] <= 1.0) {
								if (F3[i] <= 2.05) {
									if (F1[i] <= 0.732) {
										if (F5[i] <= 27.5) {
											if (F6[i] <= 41.0) {
												if (F2[i] <= 0.355) {
													if (F10[i] <= 9.125) {
														if (F0[i] <= 6.95) {
															Class[1]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F0[i] <= 7.8) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F7[i] <= 0.996) {
													Class[1]++;
												} else {
													Class[2]++;
												}
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F9[i] <= 0.545) {
											Class[1]++;
										} else {
											Class[2]++;
										}
									}
								} else {
									if (F6[i] <= 57.5) {
										if (F6[i] <= 55.0) {
											if (F2[i] <= 0.195) {
												if (F9[i] <= 0.485) {
													if (F8[i] <= 3.33) {
														Class[3]++;
													} else {
														Class[1]++;
													}
												} else {
													if (F0[i] <= 8.15) {
														if (F4[i] <= 0.064) {
															Class[0]++;
														} else {
															Class[2]++;
														}
													} else {
														if (F0[i] <= 8.7) {
															if (F7[i] <= 0.997) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[3]++;
														}
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F2[i] <= 0.315) {
											Class[2]++;
										} else {
											if (F10[i] <= 9.75) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								}
							} else {
								Class[3]++;
							}
						} else {
							Class[3]++;
						}
					} else {
						if (F9[i] <= 0.99) {
							if (F4[i] <= 0.098) {
								if (F9[i] <= 0.665) {
									if (F6[i] <= 54.5) {
										if (F6[i] <= 38.5) {
											if (F10[i] <= 9.05) {
												Class[3]++;
											} else {
												if (F1[i] <= 0.575) {
													if (F4[i] <= 0.078) {
														if (F4[i] <= 0.069) {
															if (F6[i] <= 28.5) {
																if (F0[i] <= 6.45) {
																	Class[2]++;
																} else {
																	Class[3]++;
																}
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													} else {
														if (F4[i] <= 0.088) {
															if (F0[i] <= 7.2) {
																Class[3]++;
															} else {
																Class[4]++;
															}
														} else {
															Class[3]++;
														}
													}
												} else {
													Class[2]++;
												}
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F3[i] <= 2.15) {
											if (F3[i] <= 1.65) {
												Class[2]++;
											} else {
												if (F2[i] <= 0.205) {
													if (F4[i] <= 0.082) {
														Class[2]++;
													} else {
														Class[1]++;
													}
												} else {
													Class[3]++;
												}
											}
										} else {
											Class[2]++;
										}
									}
								} else {
									if (F10[i] <= 9.55) {
										if (F3[i] <= 1.85) {
											if (F4[i] <= 0.071) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F2[i] <= 0.065) {
											Class[2]++;
										} else {
											if (F0[i] <= 7.95) {
												Class[3]++;
											} else {
												if (F1[i] <= 0.315) {
													Class[3]++;
												} else {
													if (F2[i] <= 0.41) {
														Class[4]++;
													} else {
														Class[2]++;
													}
												}
											}
										}
									}
								}
							} else {
								if (F7[i] <= 0.997) {
									Class[2]++;
								} else {
									if (F8[i] <= 3.215) {
										if (F5[i] <= 8.0) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									} else {
										if (F10[i] <= 9.7) {
											Class[3]++;
										} else {
											if (F1[i] <= 0.425) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									}
								}
							}
						} else {
							if (F7[i] <= 0.999) {
								if (F2[i] <= 0.13) {
									Class[1]++;
								} else {
									if (F10[i] <= 9.15) {
										if (F4[i] <= 0.418) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									} else {
										if (F9[i] <= 1.6) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									}
								}
							} else {
								Class[1]++;
							}
						}
					}
				} else {
					if (F4[i] <= 0.067) {
						Class[3]++;
					} else {
						if (F2[i] <= 0.235) {
							if (F4[i] <= 0.098) {
								if (F6[i] <= 100.0) {
									if (F7[i] <= 0.997) {
										if (F4[i] <= 0.089) {
											if (F1[i] <= 0.595) {
												Class[3]++;
											} else {
												if (F7[i] <= 0.997) {
													Class[2]++;
												} else {
													if (F5[i] <= 17.5) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											}
										} else {
											Class[3]++;
										}
									} else {
										Class[2]++;
									}
								} else {
									Class[2]++;
								}
							} else {
								Class[2]++;
							}
						} else {
							if (F0[i] <= 7.45) {
								if (F5[i] <= 17.5) {
									if (F9[i] <= 0.57) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								} else {
									Class[2]++;
								}
							} else {
								Class[2]++;
							}
						}
					}
				}
			} else {
				if (F10[i] <= 11.45) {
					if (F6[i] <= 88.5) {
						if (F0[i] <= 9.8) {
							if (F2[i] <= 0.585) {
								if (F10[i] <= 11.25) {
									if (F0[i] <= 6.15) {
										if (F3[i] <= 1.95) {
											if (F4[i] <= 0.073) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F1[i] <= 0.913) {
											if (F10[i] <= 11.15) {
												if (F2[i] <= 0.295) {
													if (F9[i] <= 0.545) {
														if (F6[i] <= 66.5) {
															if (F10[i] <= 11.05) {
																if (F4[i] <= 0.08) {
																	if (F2[i] <= 0.08) {
																		Class[3]++;
																	} else {
																		if (F1[i] <= 0.475) {
																			if (F5[i] <= 19.0) {
																				Class[3]++;
																			} else {
																				Class[1]++;
																			}
																		} else {
																			Class[2]++;
																		}
																	}
																} else {
																	Class[2]++;
																}
															} else {
																Class[1]++;
															}
														} else {
															if (F5[i] <= 23.5) {
																Class[2]++;
															} else {
																Class[3]++;
															}
														}
													} else {
														if (F4[i] <= 0.073) {
															if (F8[i] <= 3.24) {
																Class[3]++;
															} else {
																if (F0[i] <= 6.7) {
																	if (F5[i] <= 8.5) {
																		Class[1]++;
																	} else {
																		Class[3]++;
																	}
																} else {
																	if (F1[i] <= 0.84) {
																		if (F5[i] <= 8.0) {
																			if (F0[i] <= 7.55) {
																				Class[1]++;
																			} else {
																				Class[4]++;
																			}
																		} else {
																			if (F1[i] <= 0.688) {
																				Class[2]++;
																			} else {
																				if (F9[i] <= 0.64) {
																					Class[3]++;
																				} else {
																					Class[2]++;
																				}
																			}
																		}
																	} else {
																		Class[1]++;
																	}
																}
															}
														} else {
															if (F5[i] <= 15.5) {
																if (F9[i] <= 0.72) {
																	if (F9[i] <= 0.575) {
																		if (F3[i] <= 1.85) {
																			Class[4]++;
																		} else {
																			if (F3[i] <= 2.9) {
																				Class[3]++;
																			} else {
																				Class[2]++;
																			}
																		}
																	} else {
																		if (F2[i] <= 0.18) {
																			if (F2[i] <= 0.15) {
																				if (F3[i] <= 2.45) {
																					if (F3[i] <= 2.05) {
																						Class[2]++;
																					} else {
																						Class[3]++;
																					}
																				} else {
																					if (F4[i] <= 0.105) {
																						Class[2]++;
																					} else {
																						Class[1]++;
																					}
																				}
																			} else {
																				Class[3]++;
																			}
																		} else {
																			Class[2]++;
																		}
																	}
																} else {
																	Class[3]++;
																}
															} else {
																if (F4[i] <= 0.113) {
																	if (F2[i] <= 0.225) {
																		Class[3]++;
																	} else {
																		if (F7[i] <= 0.996) {
																			Class[2]++;
																		} else {
																			Class[3]++;
																		}
																	}
																} else {
																	Class[2]++;
																}
															}
														}
													}
												} else {
													if (F9[i] <= 0.565) {
														Class[3]++;
													} else {
														if (F7[i] <= 0.997) {
															if (F5[i] <= 42.5) {
																if (F2[i] <= 0.505) {
																	if (F2[i] <= 0.445) {
																		Class[3]++;
																	} else {
																		if (F4[i] <= 0.103) {
																			Class[3]++;
																		} else {
																			Class[2]++;
																		}
																	}
																} else {
																	if (F7[i] <= 0.996) {
																		Class[3]++;
																	} else {
																		Class[2]++;
																	}
																}
															} else {
																Class[4]++;
															}
														} else {
															if (F9[i] <= 0.59) {
																Class[1]++;
															} else {
																if (F8[i] <= 3.335) {
																	if (F5[i] <= 10.0) {
																		if (F10[i] <= 10.95) {
																			Class[3]++;
																		} else {
																			if (F2[i] <= 0.465) {
																				Class[4]++;
																			} else {
																				Class[3]++;
																			}
																		}
																	} else {
																		if (F6[i] <= 54.5) {
																			if (F8[i] <= 3.28) {
																				if (F9[i] <= 0.655) {
																					Class[2]++;
																				} else {
																					if (F9[i] <= 0.68) {
																						Class[3]++;
																					} else {
																						Class[4]++;
																					}
																				}
																			} else {
																				Class[4]++;
																			}
																		} else {
																			Class[2]++;
																		}
																	}
																} else {
																	if (F1[i] <= 0.405) {
																		if (F7[i] <= 0.998) {
																			Class[2]++;
																		} else {
																			if (F4[i] <= 0.08) {
																				Class[3]++;
																			} else {
																				Class[4]++;
																			}
																		}
																	} else {
																		Class[3]++;
																	}
																}
															}
														}
													}
												}
											} else {
												if (F2[i] <= 0.15) {
													Class[1]++;
												} else {
													if (F6[i] <= 15.5) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											}
										} else {
											if (F5[i] <= 9.0) {
												if (F5[i] <= 4.0) {
													Class[2]++;
												} else {
													if (F7[i] <= 0.996) {
														Class[1]++;
													} else {
														Class[0]++;
													}
												}
											} else {
												Class[2]++;
											}
										}
									}
								} else {
									if (F5[i] <= 5.5) {
										if (F2[i] <= 0.4) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										Class[3]++;
									}
								}
							} else {
								Class[2]++;
							}
						} else {
							if (F8[i] <= 3.05) {
								if (F6[i] <= 32.0) {
									if (F2[i] <= 0.58) {
										if (F8[i] <= 2.95) {
											Class[2]++;
										} else {
											Class[4]++;
										}
									} else {
										Class[2]++;
									}
								} else {
									Class[4]++;
								}
							} else {
								if (F8[i] <= 3.115) {
									if (F8[i] <= 3.095) {
										if (F9[i] <= 0.55) {
											Class[2]++;
										} else {
											if (F9[i] <= 0.855) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F1[i] <= 0.36) {
										if (F5[i] <= 29.5) {
											if (F10[i] <= 10.5) {
												if (F7[i] <= 0.997) {
													Class[4]++;
												} else {
													Class[3]++;
												}
											} else {
												if (F8[i] <= 3.255) {
													if (F3[i] <= 1.75) {
														if (F4[i] <= 0.058) {
															Class[4]++;
														} else {
															Class[3]++;
														}
													} else {
														Class[4]++;
													}
												} else {
													if (F9[i] <= 0.8) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											Class[4]++;
										}
									} else {
										if (F2[i] <= 0.425) {
											if (F5[i] <= 8.5) {
												Class[1]++;
											} else {
												Class[3]++;
											}
										} else {
											if (F6[i] <= 19.5) {
												Class[2]++;
											} else {
												if (F0[i] <= 10.2) {
													Class[2]++;
												} else {
													if (F7[i] <= 0.998) {
														if (F2[i] <= 0.47) {
															Class[1]++;
														} else {
															Class[2]++;
														}
													} else {
														if (F4[i] <= 0.09) {
															if (F6[i] <= 54.5) {
																Class[3]++;
															} else {
																if (F0[i] <= 12.0) {
																	Class[2]++;
																} else {
																	Class[3]++;
																}
															}
														} else {
															if (F6[i] <= 21.5) {
																Class[3]++;
															} else {
																if (F6[i] <= 35.5) {
																	Class[4]++;
																} else {
																	Class[3]++;
																}
															}
														}
													}
												}
											}
										}
									}
								}
							}
						}
					} else {
						if (F9[i] <= 1.425) {
							Class[2]++;
						} else {
							Class[3]++;
						}
					}
				} else {
					if (F2[i] <= 0.305) {
						if (F5[i] <= 8.5) {
							if (F1[i] <= 0.69) {
								Class[1]++;
							} else {
								Class[2]++;
							}
						} else {
							if (F3[i] <= 2.4) {
								Class[3]++;
							} else {
								if (F2[i] <= 0.065) {
									Class[4]++;
								} else {
									Class[3]++;
								}
							}
						}
					} else {
						if (F10[i] <= 12.45) {
							if (F9[i] <= 0.625) {
								if (F4[i] <= 0.059) {
									Class[1]++;
								} else {
									if (F1[i] <= 0.31) {
										Class[2]++;
									} else {
										if (F3[i] <= 5.4) {
											if (F7[i] <= 0.996) {
												if (F3[i] <= 2.55) {
													Class[4]++;
												} else {
													Class[3]++;
												}
											} else {
												Class[3]++;
											}
										} else {
											Class[4]++;
										}
									}
								}
							} else {
								if (F3[i] <= 2.3) {
									if (F7[i] <= 0.995) {
										Class[4]++;
									} else {
										if (F9[i] <= 0.745) {
											Class[3]++;
										} else {
											if (F7[i] <= 0.996) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									if (F7[i] <= 0.997) {
										if (F5[i] <= 10.5) {
											if (F7[i] <= 0.996) {
												Class[2]++;
											} else {
												Class[5]++;
											}
										} else {
											Class[4]++;
										}
									} else {
										Class[4]++;
									}
								}
							}
						} else {
							if (F10[i] <= 14.15) {
								if (F6[i] <= 19.5) {
									Class[5]++;
								} else {
									if (F2[i] <= 0.475) {
										Class[5]++;
									} else {
										if (F5[i] <= 8.5) {
											Class[3]++;
										} else {
											Class[4]++;
										}
									}
								}
							} else {
								Class[2]++;
							}
						}
					}
				}
			}
		}

		if (F1[i] <= 0.355) {
			if (F7[i] <= 0.995) {
				if (F9[i] <= 0.795) {
					if (F2[i] <= 0.455) {
						if (F4[i] <= 0.045) {
							Class[4]++;
						} else {
							if (F6[i] <= 35.5) {
								if (F2[i] <= 0.395) {
									if (F9[i] <= 0.535) {
										Class[3]++;
									} else {
										if (F7[i] <= 0.995) {
											Class[4]++;
										} else {
											if (F2[i] <= 0.36) {
												Class[3]++;
											} else {
												Class[4]++;
											}
										}
									}
								} else {
									if (F10[i] <= 12.55) {
										Class[3]++;
									} else {
										Class[4]++;
									}
								}
							} else {
								Class[3]++;
							}
						}
					} else {
						if (F4[i] <= 0.065) {
							if (F2[i] <= 0.66) {
								Class[3]++;
							} else {
								Class[4]++;
							}
						} else {
							if (F2[i] <= 0.53) {
								Class[4]++;
							} else {
								Class[5]++;
							}
						}
					}
				} else {
					Class[4]++;
				}
			} else {
				if (F0[i] <= 6.2) {
					if (F5[i] <= 9.0) {
						Class[1]++;
					} else {
						if (F7[i] <= 0.997) {
							Class[2]++;
						} else {
							Class[3]++;
						}
					}
				} else {
					if (F6[i] <= 22.5) {
						if (F7[i] <= 0.998) {
							if (F8[i] <= 3.365) {
								if (F10[i] <= 11.55) {
									if (F1[i] <= 0.25) {
										Class[3]++;
									} else {
										if (F3[i] <= 2.1) {
											if (F10[i] <= 10.3) {
												if (F7[i] <= 0.996) {
													Class[5]++;
												} else {
													Class[4]++;
												}
											} else {
												Class[3]++;
											}
										} else {
											Class[4]++;
										}
									}
								} else {
									Class[5]++;
								}
							} else {
								Class[4]++;
							}
						} else {
							if (F8[i] <= 3.18) {
								Class[3]++;
							} else {
								if (F4[i] <= 0.071) {
									if (F7[i] <= 0.998) {
										Class[4]++;
									} else {
										Class[3]++;
									}
								} else {
									if (F2[i] <= 0.615) {
										Class[2]++;
									} else {
										Class[4]++;
									}
								}
							}
						}
					} else {
						if (F6[i] <= 80.5) {
							if (F6[i] <= 30.5) {
								if (F9[i] <= 1.005) {
									if (F2[i] <= 0.64) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								} else {
									Class[4]++;
								}
							} else {
								if (F10[i] <= 10.5) {
									if (F2[i] <= 0.44) {
										if (F4[i] <= 0.081) {
											if (F0[i] <= 7.7) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F0[i] <= 8.05) {
										if (F6[i] <= 43.0) {
											Class[5]++;
										} else {
											if (F6[i] <= 58.0) {
												Class[4]++;
											} else {
												Class[2]++;
											}
										}
									} else {
										if (F4[i] <= 0.108) {
											if (F5[i] <= 28.5) {
												if (F10[i] <= 11.15) {
													Class[3]++;
												} else {
													Class[4]++;
												}
											} else {
												Class[4]++;
											}
										} else {
											Class[3]++;
										}
									}
								}
							}
						} else {
							if (F10[i] <= 10.1) {
								Class[2]++;
							} else {
								if (F4[i] <= 0.091) {
									Class[2]++;
								} else {
									Class[4]++;
								}
							}
						}
					}
				}
			}
		} else {
			if (F9[i] <= 0.555) {
				if (F8[i] <= 3.485) {
					if (F7[i] <= 0.995) {
						if (F9[i] <= 0.545) {
							if (F3[i] <= 2.3) {
								if (F7[i] <= 0.994) {
									Class[3]++;
								} else {
									if (F3[i] <= 1.45) {
										Class[2]++;
									} else {
										Class[3]++;
									}
								}
							} else {
								if (F1[i] <= 0.5) {
									Class[4]++;
								} else {
									Class[2]++;
								}
							}
						} else {
							Class[4]++;
						}
					} else {
						if (F10[i] <= 11.05) {
							if (F5[i] <= 30.5) {
								if (F5[i] <= 29.5) {
									if (F0[i] <= 7.95) {
										if (F3[i] <= 4.925) {
											if (F1[i] <= 0.515) {
												if (F2[i] <= 0.03) {
													Class[3]++;
												} else {
													if (F8[i] <= 3.31) {
														if (F2[i] <= 0.32) {
															if (F3[i] <= 2.75) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F6[i] <= 86.5) {
													if (F1[i] <= 0.56) {
														if (F10[i] <= 9.35) {
															Class[1]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F4[i] <= 0.096) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											if (F7[i] <= 0.997) {
												Class[3]++;
											} else {
												if (F10[i] <= 9.7) {
													Class[2]++;
												} else {
													Class[0]++;
												}
											}
										}
									} else {
										if (F1[i] <= 0.675) {
											if (F3[i] <= 3.1) {
												if (F7[i] <= 0.997) {
													if (F9[i] <= 0.455) {
														Class[2]++;
													} else {
														if (F8[i] <= 3.275) {
															if (F2[i] <= 0.265) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													if (F6[i] <= 14.5) {
														if (F2[i] <= 0.205) {
															Class[2]++;
														} else {
															Class[1]++;
														}
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F10[i] <= 10.25) {
													if (F6[i] <= 86.5) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												} else {
													Class[3]++;
												}
											}
										} else {
											if (F3[i] <= 2.05) {
												if (F8[i] <= 3.31) {
													Class[1]++;
												} else {
													Class[2]++;
												}
											} else {
												if (F8[i] <= 3.34) {
													if (F10[i] <= 9.75) {
														if (F10[i] <= 9.45) {
															if (F8[i] <= 3.21) {
																Class[2]++;
															} else {
																Class[3]++;
															}
														} else {
															Class[3]++;
														}
													} else {
														if (F9[i] <= 0.505) {
															if (F0[i] <= 8.05) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													if (F6[i] <= 29.5) {
														if (F5[i] <= 9.0) {
															if (F3[i] <= 3.75) {
																Class[0]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[1]++;
														}
													} else {
														Class[2]++;
													}
												}
											}
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								Class[2]++;
							}
						} else {
							if (F9[i] <= 0.525) {
								if (F1[i] <= 0.545) {
									Class[4]++;
								} else {
									Class[1]++;
								}
							} else {
								Class[3]++;
							}
						}
					}
				} else {
					if (F5[i] <= 23.5) {
						if (F9[i] <= 0.485) {
							if (F10[i] <= 11.65) {
								Class[1]++;
							} else {
								Class[3]++;
							}
						} else {
							if (F8[i] <= 3.585) {
								if (F4[i] <= 0.048) {
									Class[3]++;
								} else {
									if (F1[i] <= 0.84) {
										Class[2]++;
									} else {
										if (F9[i] <= 0.505) {
											Class[1]++;
										} else {
											Class[2]++;
										}
									}
								}
							} else {
								if (F6[i] <= 11.5) {
									Class[3]++;
								} else {
									if (F1[i] <= 1.102) {
										Class[1]++;
									} else {
										Class[0]++;
									}
								}
							}
						}
					} else {
						Class[3]++;
					}
				}
			} else {
				if (F3[i] <= 4.95) {
					if (F1[i] <= 0.605) {
						if (F9[i] <= 0.715) {
							if (F5[i] <= 19.5) {
								if (F10[i] <= 9.75) {
									if (F8[i] <= 3.365) {
										if (F10[i] <= 9.35) {
											if (F0[i] <= 6.7) {
												Class[2]++;
											} else {
												if (F10[i] <= 9.25) {
													Class[3]++;
												} else {
													if (F7[i] <= 0.999) {
														if (F0[i] <= 9.6) {
															if (F5[i] <= 8.0) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[3]++;
														}
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											if (F9[i] <= 0.615) {
												if (F3[i] <= 2.25) {
													if (F4[i] <= 0.073) {
														if (F5[i] <= 6.0) {
															Class[2]++;
														} else {
															Class[4]++;
														}
													} else {
														Class[3]++;
													}
												} else {
													if (F7[i] <= 0.998) {
														Class[2]++;
													} else {
														Class[4]++;
													}
												}
											} else {
												if (F9[i] <= 0.68) {
													if (F5[i] <= 6.5) {
														if (F8[i] <= 3.115) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													} else {
														if (F0[i] <= 7.75) {
															if (F7[i] <= 0.996) {
																Class[2]++;
															} else {
																Class[3]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													Class[3]++;
												}
											}
										}
									} else {
										if (F1[i] <= 0.585) {
											if (F2[i] <= 0.18) {
												if (F0[i] <= 7.05) {
													if (F2[i] <= 0.03) {
														Class[2]++;
													} else {
														if (F7[i] <= 0.996) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													}
												} else {
													Class[4]++;
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									}
								} else {
									if (F8[i] <= 3.595) {
										if (F4[i] <= 0.091) {
											if (F0[i] <= 9.35) {
												if (F7[i] <= 0.998) {
													if (F9[i] <= 0.705) {
														if (F3[i] <= 2.15) {
															if (F0[i] <= 6.15) {
																if (F10[i] <= 11.8) {
																	Class[2]++;
																} else {
																	Class[3]++;
																}
															} else {
																if (F6[i] <= 32.5) {
																	Class[3]++;
																} else {
																	if (F10[i] <= 10.25) {
																		Class[3]++;
																	} else {
																		if (F2[i] <= 0.215) {
																			if (F7[i] <= 0.995) {
																				Class[3]++;
																			} else {
																				Class[2]++;
																			}
																		} else {
																			Class[3]++;
																		}
																	}
																}
															}
														} else {
															if (F9[i] <= 0.595) {
																if (F8[i] <= 3.235) {
																	Class[2]++;
																} else {
																	if (F2[i] <= 0.11) {
																		if (F5[i] <= 11.0) {
																			Class[1]++;
																		} else {
																			Class[3]++;
																		}
																	} else {
																		Class[3]++;
																	}
																}
															} else {
																if (F0[i] <= 8.25) {
																	if (F10[i] <= 10.35) {
																		Class[4]++;
																	} else {
																		if (F10[i] <= 11.65) {
																			Class[3]++;
																		} else {
																			Class[4]++;
																		}
																	}
																} else {
																	Class[4]++;
																}
															}
														}
													} else {
														if (F2[i] <= 0.285) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													}
												} else {
													Class[2]++;
												}
											} else {
												if (F0[i] <= 12.3) {
													Class[3]++;
												} else {
													if (F5[i] <= 6.0) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											if (F1[i] <= 0.52) {
												if (F7[i] <= 0.998) {
													if (F8[i] <= 3.21) {
														Class[4]++;
													} else {
														if (F7[i] <= 0.996) {
															if (F6[i] <= 9.0) {
																Class[1]++;
															} else {
																Class[3]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													Class[2]++;
												}
											} else {
												if (F4[i] <= 0.115) {
													Class[2]++;
												} else {
													if (F9[i] <= 0.635) {
														Class[3]++;
													} else {
														Class[1]++;
													}
												}
											}
										}
									} else {
										if (F3[i] <= 1.5) {
											Class[3]++;
										} else {
											if (F7[i] <= 0.995) {
												Class[2]++;
											} else {
												if (F5[i] <= 6.0) {
													Class[1]++;
												} else {
													Class[2]++;
												}
											}
										}
									}
								}
							} else {
								if (F10[i] <= 10.65) {
									if (F6[i] <= 70.5) {
										if (F5[i] <= 32.5) {
											if (F3[i] <= 3.65) {
												if (F5[i] <= 22.5) {
													if (F7[i] <= 0.997) {
														if (F9[i] <= 0.61) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F6[i] <= 61.5) {
														if (F9[i] <= 0.645) {
															if (F3[i] <= 1.95) {
																if (F6[i] <= 52.5) {
																	Class[2]++;
																} else {
																	Class[3]++;
																}
															} else {
																Class[2]++;
															}
														} else {
															if (F7[i] <= 0.996) {
																Class[3]++;
															} else {
																Class[4]++;
															}
														}
													} else {
														Class[3]++;
													}
												}
											} else {
												Class[1]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F5[i] <= 32.5) {
											if (F6[i] <= 96.0) {
												if (F8[i] <= 3.26) {
													Class[3]++;
												} else {
													if (F10[i] <= 9.6) {
														if (F8[i] <= 3.395) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									}
								} else {
									if (F3[i] <= 2.55) {
										if (F6[i] <= 40.5) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										if (F10[i] <= 10.8) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									}
								}
							}
						} else {
							if (F10[i] <= 12.45) {
								if (F4[i] <= 0.104) {
									if (F8[i] <= 3.345) {
										if (F3[i] <= 4.05) {
											if (F4[i] <= 0.071) {
												if (F0[i] <= 8.45) {
													Class[4]++;
												} else {
													if (F4[i] <= 0.049) {
														Class[3]++;
													} else {
														if (F3[i] <= 1.95) {
															Class[3]++;
														} else {
															if (F9[i] <= 0.835) {
																Class[2]++;
															} else {
																if (F2[i] <= 0.565) {
																	Class[3]++;
																} else {
																	Class[2]++;
																}
															}
														}
													}
												}
											} else {
												if (F1[i] <= 0.51) {
													if (F6[i] <= 122.0) {
														if (F5[i] <= 28.5) {
															if (F8[i] <= 3.04) {
																if (F5[i] <= 14.0) {
																	Class[3]++;
																} else {
																	Class[4]++;
																}
															} else {
																if (F0[i] <= 10.8) {
																	Class[3]++;
																} else {
																	if (F6[i] <= 37.5) {
																		if (F5[i] <= 8.5) {
																			Class[3]++;
																		} else {
																			Class[4]++;
																		}
																	} else {
																		Class[3]++;
																	}
																}
															}
														} else {
															if (F3[i] <= 2.5) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F1[i] <= 0.565) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											Class[4]++;
										}
									} else {
										if (F6[i] <= 53.5) {
											Class[3]++;
										} else {
											if (F9[i] <= 0.755) {
												Class[2]++;
											} else {
												if (F10[i] <= 10.25) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											}
										}
									}
								} else {
									if (F10[i] <= 9.65) {
										if (F9[i] <= 0.885) {
											Class[2]++;
										} else {
											if (F4[i] <= 0.132) {
												Class[2]++;
											} else {
												if (F4[i] <= 0.418) {
													if (F9[i] <= 0.98) {
														Class[3]++;
													} else {
														if (F2[i] <= 0.49) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													}
												} else {
													Class[2]++;
												}
											}
										}
									} else {
										if (F2[i] <= 0.595) {
											if (F1[i] <= 0.535) {
												if (F0[i] <= 11.25) {
													if (F8[i] <= 3.375) {
														if (F2[i] <= 0.37) {
															if (F5[i] <= 19.5) {
																Class[2]++;
															} else {
																Class[3]++;
															}
														} else {
															Class[3]++;
														}
													} else {
														Class[0]++;
													}
												} else {
													Class[1]++;
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[4]++;
										}
									}
								}
							} else {
								if (F9[i] <= 0.83) {
									if (F2[i] <= 0.36) {
										if (F2[i] <= 0.085) {
											if (F7[i] <= 0.992) {
												Class[5]++;
											} else {
												Class[4]++;
											}
										} else {
											Class[5]++;
										}
									} else {
										if (F4[i] <= 0.096) {
											if (F6[i] <= 22.0) {
												Class[3]++;
											} else {
												Class[4]++;
											}
										} else {
											Class[2]++;
										}
									}
								} else {
									Class[4]++;
								}
							}
						}
					} else {
						if (F7[i] <= 0.996) {
							if (F9[i] <= 0.625) {
								if (F5[i] <= 12.5) {
									if (F10[i] <= 11.1) {
										if (F8[i] <= 3.475) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										if (F0[i] <= 6.05) {
											Class[2]++;
										} else {
											Class[1]++;
										}
									}
								} else {
									if (F9[i] <= 0.595) {
										Class[3]++;
									} else {
										if (F6[i] <= 56.5) {
											Class[3]++;
										} else {
											if (F3[i] <= 1.8) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								}
							} else {
								if (F10[i] <= 12.35) {
									if (F6[i] <= 89.0) {
										if (F10[i] <= 11.6) {
											Class[3]++;
										} else {
											if (F9[i] <= 0.725) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										}
									} else {
										Class[2]++;
									}
								} else {
									Class[5]++;
								}
							}
						} else {
							if (F10[i] <= 10.35) {
								if (F7[i] <= 1.0) {
									if (F6[i] <= 17.0) {
										if (F9[i] <= 0.585) {
											Class[2]++;
										} else {
											if (F1[i] <= 0.743) {
												Class[3]++;
											} else {
												Class[1]++;
											}
										}
									} else {
										if (F8[i] <= 2.945) {
											Class[3]++;
										} else {
											if (F3[i] <= 4.0) {
												if (F4[i] <= 0.098) {
													if (F2[i] <= 0.265) {
														if (F4[i] <= 0.095) {
															if (F0[i] <= 7.65) {
																if (F3[i] <= 2.05) {
																	if (F8[i] <= 3.26) {
																		Class[1]++;
																	} else {
																		if (F2[i] <= 0.07) {
																			Class[2]++;
																		} else {
																			if (F3[i] <= 1.85) {
																				if (F9[i] <= 0.68) {
																					Class[2]++;
																				} else {
																					Class[3]++;
																				}
																			} else {
																				Class[3]++;
																			}
																		}
																	}
																} else {
																	if (F7[i] <= 0.997) {
																		Class[2]++;
																	} else {
																		Class[3]++;
																	}
																}
															} else {
																Class[2]++;
															}
														} else {
															if (F0[i] <= 9.15) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													} else {
														if (F8[i] <= 3.07) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													}
												} else {
													Class[2]++;
												}
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								if (F10[i] <= 11.1) {
									if (F9[i] <= 0.61) {
										if (F7[i] <= 0.996) {
											Class[2]++;
										} else {
											if (F3[i] <= 1.65) {
												Class[1]++;
											} else {
												if (F3[i] <= 3.6) {
													if (F1[i] <= 0.947) {
														Class[3]++;
													} else {
														Class[1]++;
													}
												} else {
													Class[1]++;
												}
											}
										}
									} else {
										if (F0[i] <= 9.25) {
											if (F7[i] <= 0.997) {
												if (F5[i] <= 12.0) {
													Class[3]++;
												} else {
													if (F5[i] <= 21.5) {
														Class[4]++;
													} else {
														Class[2]++;
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									}
								} else {
									if (F8[i] <= 3.16) {
										Class[4]++;
									} else {
										Class[2]++;
									}
								}
							}
						}
					}
				} else {
					if (F3[i] <= 8.45) {
						if (F9[i] <= 0.57) {
							Class[1]++;
						} else {
							if (F3[i] <= 6.575) {
								if (F7[i] <= 0.999) {
									Class[2]++;
								} else {
									if (F2[i] <= 0.735) {
										if (F6[i] <= 29.5) {
											Class[3]++;
										} else {
											if (F4[i] <= 0.098) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										}
									} else {
										Class[2]++;
									}
								}
							} else {
								Class[2]++;
							}
						}
					} else {
						Class[3]++;
					}
				}
			}
		}

		if (F4[i] <= 0.068) {
			if (F2[i] <= 0.275) {
				if (F6[i] <= 47.5) {
					if (F7[i] <= 0.995) {
						if (F9[i] <= 0.545) {
							if (F1[i] <= 0.725) {
								if (F10[i] <= 10.85) {
									if (F6[i] <= 23.5) {
										Class[4]++;
									} else {
										Class[2]++;
									}
								} else {
									Class[2]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F0[i] <= 6.95) {
								if (F2[i] <= 0.015) {
									if (F7[i] <= 0.995) {
										Class[3]++;
									} else {
										if (F4[i] <= 0.057) {
											Class[2]++;
										} else {
											Class[1]++;
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								if (F3[i] <= 1.85) {
									Class[4]++;
								} else {
									if (F10[i] <= 12.4) {
										Class[3]++;
									} else {
										Class[5]++;
									}
								}
							}
						}
					} else {
						if (F0[i] <= 7.3) {
							if (F9[i] <= 0.645) {
								if (F0[i] <= 6.65) {
									if (F5[i] <= 13.5) {
										if (F8[i] <= 3.56) {
											Class[2]++;
										} else {
											Class[1]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F5[i] <= 3.5) {
										Class[2]++;
									} else {
										Class[1]++;
									}
								}
							} else {
								if (F5[i] <= 6.0) {
									Class[2]++;
								} else {
									Class[3]++;
								}
							}
						} else {
							if (F0[i] <= 8.9) {
								if (F1[i] <= 0.875) {
									Class[2]++;
								} else {
									Class[3]++;
								}
							} else {
								Class[3]++;
							}
						}
					}
				} else {
					if (F10[i] <= 12.2) {
						if (F0[i] <= 5.3) {
							Class[1]++;
						} else {
							if (F3[i] <= 1.75) {
								Class[2]++;
							} else {
								if (F10[i] <= 10.275) {
									if (F6[i] <= 55.5) {
										Class[1]++;
									} else {
										Class[2]++;
									}
								} else {
									if (F5[i] <= 18.0) {
										Class[2]++;
									} else {
										Class[3]++;
									}
								}
							}
						}
					} else {
						if (F6[i] <= 88.5) {
							if (F0[i] <= 6.1) {
								Class[5]++;
							} else {
								Class[4]++;
							}
						} else {
							if (F9[i] <= 0.74) {
								Class[2]++;
							} else {
								Class[4]++;
							}
						}
					}
				}
			} else {
				if (F6[i] <= 10.5) {
					Class[4]++;
				} else {
					if (F5[i] <= 30.0) {
						if (F7[i] <= 0.997) {
							if (F3[i] <= 1.65) {
								Class[3]++;
							} else {
								if (F0[i] <= 6.85) {
									if (F8[i] <= 3.33) {
										Class[1]++;
									} else {
										Class[3]++;
									}
								} else {
									if (F4[i] <= 0.053) {
										Class[3]++;
									} else {
										if (F6[i] <= 28.5) {
											if (F5[i] <= 8.5) {
												if (F0[i] <= 8.85) {
													Class[3]++;
												} else {
													Class[4]++;
												}
											} else {
												Class[4]++;
											}
										} else {
											if (F7[i] <= 0.996) {
												if (F2[i] <= 0.325) {
													Class[5]++;
												} else {
													Class[3]++;
												}
											} else {
												if (F4[i] <= 0.055) {
													Class[4]++;
												} else {
													Class[2]++;
												}
											}
										}
									}
								}
							}
						} else {
							if (F0[i] <= 11.65) {
								if (F4[i] <= 0.055) {
									if (F0[i] <= 10.6) {
										Class[2]++;
									} else {
										Class[4]++;
									}
								} else {
									if (F3[i] <= 2.15) {
										Class[3]++;
									} else {
										if (F6[i] <= 28.0) {
											if (F6[i] <= 15.0) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										} else {
											if (F6[i] <= 81.5) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								}
							} else {
								if (F7[i] <= 0.998) {
									Class[4]++;
								} else {
									Class[2]++;
								}
							}
						}
					} else {
						if (F10[i] <= 9.95) {
							Class[2]++;
						} else {
							if (F1[i] <= 0.215) {
								if (F3[i] <= 1.8) {
									Class[3]++;
								} else {
									Class[2]++;
								}
							} else {
								if (F0[i] <= 9.0) {
									if (F6[i] <= 69.5) {
										if (F4[i] <= 0.061) {
											Class[3]++;
										} else {
											Class[4]++;
										}
									} else {
										Class[4]++;
									}
								} else {
									Class[3]++;
								}
							}
						}
					}
				}
			}
		} else {
			if (F10[i] <= 9.85) {
				if (F9[i] <= 0.575) {
					if (F7[i] <= 0.997) {
						if (F7[i] <= 0.997) {
							if (F3[i] <= 1.35) {
								Class[0]++;
							} else {
								if (F5[i] <= 11.5) {
									if (F4[i] <= 0.08) {
										if (F8[i] <= 3.24) {
											if (F1[i] <= 0.55) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F9[i] <= 0.495) {
											Class[1]++;
										} else {
											Class[2]++;
										}
									}
								} else {
									if (F3[i] <= 1.95) {
										if (F9[i] <= 0.535) {
											Class[2]++;
										} else {
											if (F2[i] <= 0.09) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									} else {
										if (F10[i] <= 9.75) {
											if (F1[i] <= 0.44) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										} else {
											if (F5[i] <= 14.5) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								}
							}
						} else {
							if (F8[i] <= 3.22) {
								Class[2]++;
							} else {
								if (F4[i] <= 0.074) {
									Class[4]++;
								} else {
									Class[3]++;
								}
							}
						}
					} else {
						if (F6[i] <= 57.5) {
							if (F1[i] <= 0.365) {
								Class[3]++;
							} else {
								if (F6[i] <= 42.0) {
									if (F0[i] <= 12.75) {
										Class[2]++;
									} else {
										Class[3]++;
									}
								} else {
									if (F10[i] <= 9.35) {
										Class[1]++;
									} else {
										if (F2[i] <= 0.2) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									}
								}
							}
						} else {
							if (F3[i] <= 9.3) {
								Class[2]++;
							} else {
								Class[3]++;
							}
						}
					}
				} else {
					if (F3[i] <= 1.35) {
						if (F0[i] <= 8.35) {
							Class[1]++;
						} else {
							Class[2]++;
						}
					} else {
						if (F4[i] <= 0.098) {
							if (F8[i] <= 3.535) {
								if (F3[i] <= 2.05) {
									if (F5[i] <= 9.0) {
										Class[3]++;
									} else {
										if (F10[i] <= 9.75) {
											if (F10[i] <= 9.55) {
												if (F1[i] <= 0.64) {
													if (F7[i] <= 0.996) {
														Class[2]++;
													} else {
														if (F6[i] <= 81.5) {
															if (F7[i] <= 0.998) {
																if (F0[i] <= 6.5) {
																	if (F4[i] <= 0.083) {
																		Class[3]++;
																	} else {
																		Class[2]++;
																	}
																} else {
																	Class[3]++;
																}
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													if (F3[i] <= 1.9) {
														Class[2]++;
													} else {
														Class[1]++;
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											if (F0[i] <= 7.45) {
												if (F3[i] <= 1.85) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									if (F9[i] <= 0.695) {
										if (F5[i] <= 9.0) {
											if (F6[i] <= 12.0) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											if (F4[i] <= 0.095) {
												if (F1[i] <= 0.585) {
													if (F9[i] <= 0.585) {
														Class[3]++;
													} else {
														if (F6[i] <= 89.0) {
															if (F1[i] <= 0.52) {
																if (F7[i] <= 0.998) {
																	Class[2]++;
																} else {
																	if (F1[i] <= 0.495) {
																		Class[3]++;
																	} else {
																		Class[4]++;
																	}
																}
															} else {
																Class[2]++;
															}
														} else {
															Class[2]++;
														}
													}
												} else {
													if (F9[i] <= 0.595) {
														Class[1]++;
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F3[i] <= 2.8) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											}
										}
									} else {
										if (F1[i] <= 0.235) {
											Class[4]++;
										} else {
											if (F3[i] <= 5.55) {
												if (F8[i] <= 3.045) {
													if (F1[i] <= 0.34) {
														Class[5]++;
													} else {
														Class[2]++;
													}
												} else {
													if (F7[i] <= 0.998) {
														if (F9[i] <= 0.72) {
															Class[3]++;
														} else {
															if (F4[i] <= 0.08) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													} else {
														Class[3]++;
													}
												}
											} else {
												Class[2]++;
											}
										}
									}
								}
							} else {
								if (F3[i] <= 2.85) {
									Class[2]++;
								} else {
									Class[3]++;
								}
							}
						} else {
							if (F3[i] <= 11.75) {
								if (F3[i] <= 2.05) {
									if (F0[i] <= 8.8) {
										if (F1[i] <= 0.59) {
											if (F5[i] <= 3.5) {
												Class[3]++;
											} else {
												if (F8[i] <= 3.305) {
													if (F8[i] <= 3.075) {
														Class[2]++;
													} else {
														if (F10[i] <= 9.2) {
															Class[3]++;
														} else {
															if (F7[i] <= 0.998) {
																Class[2]++;
															} else {
																Class[3]++;
															}
														}
													}
												} else {
													Class[3]++;
												}
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F10[i] <= 9.35) {
											if (F4[i] <= 0.342) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[3]++;
										}
									}
								} else {
									if (F9[i] <= 0.605) {
										if (F8[i] <= 3.26) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										Class[2]++;
									}
								}
							} else {
								Class[3]++;
							}
						}
					}
				}
			} else {
				if (F10[i] <= 11.45) {
					if (F1[i] <= 0.835) {
						if (F4[i] <= 0.072) {
							if (F2[i] <= 0.455) {
								if (F0[i] <= 6.95) {
									if (F2[i] <= 0.05) {
										if (F3[i] <= 2.0) {
											Class[1]++;
										} else {
											Class[2]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									Class[2]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F2[i] <= 0.53) {
								if (F1[i] <= 0.595) {
									if (F10[i] <= 10.25) {
										if (F5[i] <= 33.5) {
											if (F2[i] <= 0.515) {
												if (F1[i] <= 0.565) {
													if (F9[i] <= 0.485) {
														Class[2]++;
													} else {
														if (F6[i] <= 16.0) {
															Class[1]++;
														} else {
															if (F9[i] <= 0.6) {
																if (F1[i] <= 0.435) {
																	Class[2]++;
																} else {
																	Class[3]++;
																}
															} else {
																Class[3]++;
															}
														}
													}
												} else {
													if (F1[i] <= 0.58) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[0]++;
										}
									} else {
										if (F9[i] <= 0.545) {
											if (F2[i] <= 0.315) {
												if (F0[i] <= 6.15) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												if (F2[i] <= 0.465) {
													Class[3]++;
												} else {
													if (F5[i] <= 5.5) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											}
										} else {
											if (F6[i] <= 71.0) {
												if (F4[i] <= 0.097) {
													if (F3[i] <= 1.85) {
														if (F6[i] <= 31.0) {
															if (F5[i] <= 16.0) {
																if (F1[i] <= 0.405) {
																	Class[4]++;
																} else {
																	if (F2[i] <= 0.115) {
																		Class[4]++;
																	} else {
																		Class[3]++;
																	}
																}
															} else {
																Class[5]++;
															}
														} else {
															if (F10[i] <= 10.55) {
																Class[2]++;
															} else {
																if (F6[i] <= 61.5) {
																	Class[3]++;
																} else {
																	Class[2]++;
																}
															}
														}
													} else {
														if (F1[i] <= 0.265) {
															if (F8[i] <= 3.27) {
																if (F4[i] <= 0.077) {
																	Class[2]++;
																} else {
																	Class[4]++;
																}
															} else {
																Class[2]++;
															}
														} else {
															if (F5[i] <= 28.5) {
																if (F5[i] <= 23.0) {
																	Class[3]++;
																} else {
																	if (F2[i] <= 0.42) {
																		Class[3]++;
																	} else {
																		if (F10[i] <= 10.7) {
																			Class[3]++;
																		} else {
																			Class[4]++;
																		}
																	}
																}
															} else {
																if (F10[i] <= 10.45) {
																	Class[2]++;
																} else {
																	if (F0[i] <= 9.6) {
																		Class[3]++;
																	} else {
																		Class[2]++;
																	}
																}
															}
														}
													}
												} else {
													if (F9[i] <= 0.735) {
														if (F2[i] <= 0.235) {
															if (F4[i] <= 0.122) {
																Class[3]++;
															} else {
																Class[1]++;
															}
														} else {
															if (F7[i] <= 0.996) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													} else {
														Class[3]++;
													}
												}
											} else {
												if (F0[i] <= 7.9) {
													Class[2]++;
												} else {
													if (F10[i] <= 10.85) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												}
											}
										}
									}
								} else {
									if (F6[i] <= 78.0) {
										if (F8[i] <= 3.145) {
											if (F6[i] <= 50.5) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											if (F4[i] <= 0.083) {
												if (F8[i] <= 3.51) {
													if (F2[i] <= 0.29) {
														if (F8[i] <= 3.295) {
															if (F8[i] <= 3.275) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														} else {
															Class[3]++;
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F0[i] <= 6.45) {
														Class[2]++;
													} else {
														Class[4]++;
													}
												}
											} else {
												if (F6[i] <= 62.0) {
													if (F6[i] <= 47.5) {
														if (F5[i] <= 14.5) {
															if (F6[i] <= 37.0) {
																if (F10[i] <= 10.6) {
																	Class[2]++;
																} else {
																	if (F5[i] <= 7.5) {
																		if (F10[i] <= 10.85) {
																			Class[3]++;
																		} else {
																			Class[2]++;
																		}
																	} else {
																		Class[3]++;
																	}
																}
															} else {
																Class[3]++;
															}
														} else {
															if (F3[i] <= 2.9) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													} else {
														if (F2[i] <= 0.355) {
															Class[2]++;
														} else {
															Class[1]++;
														}
													}
												} else {
													Class[3]++;
												}
											}
										}
									} else {
										Class[2]++;
									}
								}
							} else {
								if (F6[i] <= 26.5) {
									if (F2[i] <= 0.64) {
										if (F9[i] <= 0.73) {
											if (F8[i] <= 3.385) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										} else {
											Class[4]++;
										}
									} else {
										Class[2]++;
									}
								} else {
									if (F2[i] <= 0.595) {
										if (F5[i] <= 30.0) {
											if (F8[i] <= 3.085) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[4]++;
										}
									} else {
										if (F4[i] <= 0.094) {
											if (F2[i] <= 0.655) {
												if (F8[i] <= 3.03) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												Class[4]++;
											}
										} else {
											Class[4]++;
										}
									}
								}
							}
						}
					} else {
						if (F6[i] <= 28.0) {
							if (F10[i] <= 11.1) {
								if (F4[i] <= 0.075) {
									Class[3]++;
								} else {
									if (F6[i] <= 19.5) {
										if (F7[i] <= 0.996) {
											Class[2]++;
										} else {
											if (F6[i] <= 13.5) {
												if (F0[i] <= 7.7) {
													Class[1]++;
												} else {
													Class[0]++;
												}
											} else {
												Class[0]++;
											}
										}
									} else {
										Class[1]++;
									}
								}
							} else {
								Class[1]++;
							}
						} else {
							if (F4[i] <= 0.078) {
								Class[3]++;
							} else {
								Class[2]++;
							}
						}
					}
				} else {
					if (F8[i] <= 3.375) {
						if (F3[i] <= 1.95) {
							if (F9[i] <= 0.71) {
								Class[5]++;
							} else {
								Class[3]++;
							}
						} else {
							if (F2[i] <= 0.315) {
								Class[3]++;
							} else {
								if (F6[i] <= 9.5) {
									Class[3]++;
								} else {
									if (F0[i] <= 10.7) {
										if (F4[i] <= 0.124) {
											if (F5[i] <= 27.5) {
												if (F9[i] <= 0.715) {
													Class[4]++;
												} else {
													if (F6[i] <= 18.0) {
														if (F0[i] <= 9.55) {
															Class[5]++;
														} else {
															Class[3]++;
														}
													} else {
														if (F0[i] <= 8.4) {
															if (F8[i] <= 3.345) {
																Class[4]++;
															} else {
																Class[5]++;
															}
														} else {
															Class[4]++;
														}
													}
												}
											} else {
												if (F6[i] <= 47.0) {
													Class[3]++;
												} else {
													if (F6[i] <= 53.5) {
														Class[4]++;
													} else {
														if (F2[i] <= 0.39) {
															Class[4]++;
														} else {
															Class[3]++;
														}
													}
												}
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F6[i] <= 59.0) {
											if (F9[i] <= 0.685) {
												Class[3]++;
											} else {
												if (F9[i] <= 0.84) {
													Class[5]++;
												} else {
													Class[3]++;
												}
											}
										} else {
											Class[2]++;
										}
									}
								}
							}
						}
					} else {
						if (F4[i] <= 0.084) {
							if (F4[i] <= 0.071) {
								if (F9[i] <= 0.67) {
									Class[3]++;
								} else {
									Class[4]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F6[i] <= 35.5) {
								if (F5[i] <= 6.0) {
									if (F10[i] <= 12.15) {
										Class[1]++;
									} else {
										Class[4]++;
									}
								} else {
									Class[4]++;
								}
							} else {
								Class[3]++;
							}
						}
					}
				}
			}
		}

		if (F9[i] <= 0.545) {
			if (F7[i] <= 0.995) {
				if (F2[i] <= 0.275) {
					if (F7[i] <= 0.993) {
						Class[3]++;
					} else {
						if (F7[i] <= 0.995) {
							if (F1[i] <= 0.837) {
								if (F6[i] <= 10.0) {
									Class[1]++;
								} else {
									Class[2]++;
								}
							} else {
								if (F7[i] <= 0.994) {
									Class[3]++;
								} else {
									Class[1]++;
								}
							}
						} else {
							if (F1[i] <= 0.743) {
								if (F2[i] <= 0.23) {
									Class[3]++;
								} else {
									Class[2]++;
								}
							} else {
								if (F10[i] <= 11.15) {
									Class[0]++;
								} else {
									Class[1]++;
								}
							}
						}
					}
				} else {
					if (F3[i] <= 2.8) {
						if (F5[i] <= 5.5) {
							if (F2[i] <= 0.46) {
								Class[4]++;
							} else {
								Class[3]++;
							}
						} else {
							Class[3]++;
						}
					} else {
						Class[4]++;
					}
				}
			} else {
				if (F4[i] <= 0.076) {
					if (F4[i] <= 0.071) {
						if (F9[i] <= 0.435) {
							Class[3]++;
						} else {
							if (F6[i] <= 140.5) {
								if (F9[i] <= 0.51) {
									Class[2]++;
								} else {
									if (F6[i] <= 34.5) {
										if (F6[i] <= 23.0) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										Class[2]++;
									}
								}
							} else {
								Class[3]++;
							}
						}
					} else {
						if (F9[i] <= 0.535) {
							if (F7[i] <= 0.999) {
								if (F0[i] <= 8.5) {
									if (F7[i] <= 0.995) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								} else {
									if (F1[i] <= 0.352) {
										Class[3]++;
									} else {
										if (F1[i] <= 0.682) {
											Class[4]++;
										} else {
											Class[1]++;
										}
									}
								}
							} else {
								Class[3]++;
							}
						} else {
							Class[3]++;
						}
					}
				} else {
					if (F1[i] <= 0.455) {
						if (F4[i] <= 0.082) {
							if (F7[i] <= 0.996) {
								Class[3]++;
							} else {
								if (F0[i] <= 7.65) {
									Class[2]++;
								} else {
									if (F9[i] <= 0.525) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								}
							}
						} else {
							if (F2[i] <= 0.645) {
								if (F3[i] <= 2.25) {
									if (F9[i] <= 0.495) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								} else {
									Class[2]++;
								}
							} else {
								Class[3]++;
							}
						}
					} else {
						if (F0[i] <= 7.95) {
							if (F8[i] <= 3.57) {
								if (F3[i] <= 4.925) {
									if (F5[i] <= 22.5) {
										Class[2]++;
									} else {
										if (F7[i] <= 0.997) {
											Class[2]++;
										} else {
											if (F6[i] <= 69.5) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									if (F3[i] <= 5.3) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								}
							} else {
								Class[1]++;
							}
						} else {
							if (F10[i] <= 9.45) {
								if (F2[i] <= 0.295) {
									Class[2]++;
								} else {
									if (F3[i] <= 2.5) {
										Class[2]++;
									} else {
										Class[1]++;
									}
								}
							} else {
								if (F2[i] <= 0.315) {
									if (F7[i] <= 0.997) {
										if (F1[i] <= 0.893) {
											Class[2]++;
										} else {
											if (F1[i] <= 1.123) {
												Class[1]++;
											} else {
												Class[2]++;
											}
										}
									} else {
										if (F4[i] <= 0.094) {
											if (F3[i] <= 2.225) {
												if (F6[i] <= 28.0) {
													Class[1]++;
												} else {
													Class[3]++;
												}
											} else {
												if (F9[i] <= 0.47) {
													Class[1]++;
												} else {
													if (F3[i] <= 2.8) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												}
											}
										} else {
											Class[2]++;
										}
									}
								} else {
									Class[3]++;
								}
							}
						}
					}
				}
			}
		} else {
			if (F10[i] <= 10.55) {
				if (F0[i] <= 9.95) {
					if (F1[i] <= 0.37) {
						if (F5[i] <= 10.0) {
							Class[2]++;
						} else {
							if (F2[i] <= 0.28) {
								if (F1[i] <= 0.315) {
									Class[4]++;
								} else {
									Class[2]++;
								}
							} else {
								if (F10[i] <= 10.45) {
									if (F1[i] <= 0.255) {
										if (F9[i] <= 0.88) {
											Class[4]++;
										} else {
											Class[3]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									Class[4]++;
								}
							}
						}
					} else {
						if (F4[i] <= 0.098) {
							if (F9[i] <= 0.805) {
								if (F2[i] <= 0.005) {
									if (F8[i] <= 3.475) {
										if (F6[i] <= 34.5) {
											Class[3]++;
										} else {
											if (F8[i] <= 3.42) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									} else {
										if (F0[i] <= 6.5) {
											Class[1]++;
										} else {
											Class[2]++;
										}
									}
								} else {
									if (F6[i] <= 91.5) {
										if (F2[i] <= 0.43) {
											if (F4[i] <= 0.095) {
												if (F1[i] <= 0.605) {
													if (F5[i] <= 31.5) {
														if (F10[i] <= 9.45) {
															if (F8[i] <= 3.305) {
																if (F2[i] <= 0.255) {
																	Class[3]++;
																} else {
																	Class[2]++;
																}
															} else {
																Class[2]++;
															}
														} else {
															if (F6[i] <= 20.0) {
																if (F7[i] <= 0.997) {
																	Class[4]++;
																} else {
																	Class[2]++;
																}
															} else {
																if (F7[i] <= 0.999) {
																	if (F4[i] <= 0.069) {
																		if (F9[i] <= 0.645) {
																			if (F1[i] <= 0.535) {
																				Class[2]++;
																			} else {
																				Class[3]++;
																			}
																		} else {
																			if (F2[i] <= 0.145) {
																				Class[3]++;
																			} else {
																				Class[4]++;
																			}
																		}
																	} else {
																		if (F5[i] <= 7.5) {
																			Class[2]++;
																		} else {
																			if (F2[i] <= 0.255) {
																				if (F3[i] <= 1.95) {
																					if (F0[i] <= 7.2) {
																						Class[3]++;
																					} else {
																						Class[4]++;
																					}
																				} else {
																					if (F6[i] <= 47.5) {
																						if (F3[i] <= 2.05) {
																							if (F0[i] <= 6.35) {
																								Class[3]++;
																							} else {
																								Class[2]++;
																							}
																						} else {
																							if (F9[i] <= 0.585) {
																								if (F10[i] <= 9.75) {
																									Class[3]++;
																								} else {
																									Class[2]++;
																								}
																							} else {
																								Class[3]++;
																							}
																						}
																					} else {
																						if (F3[i] <= 2.5) {
																							Class[2]++;
																						} else {
																							if (F6[i] <= 55.5) {
																								Class[2]++;
																							} else {
																								Class[3]++;
																							}
																						}
																					}
																				}
																			} else {
																				if (F2[i] <= 0.315) {
																					Class[3]++;
																				} else {
																					if (F6[i] <= 48.0) {
																						Class[3]++;
																					} else {
																						Class[2]++;
																					}
																				}
																			}
																		}
																	}
																} else {
																	Class[4]++;
																}
															}
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F9[i] <= 0.555) {
														if (F8[i] <= 3.3) {
															Class[2]++;
														} else {
															if (F6[i] <= 46.5) {
																if (F2[i] <= 0.165) {
																	Class[3]++;
																} else {
																	Class[2]++;
																}
															} else {
																Class[0]++;
															}
														}
													} else {
														if (F3[i] <= 2.65) {
															if (F5[i] <= 17.5) {
																if (F7[i] <= 0.996) {
																	if (F8[i] <= 3.39) {
																		Class[3]++;
																	} else {
																		Class[2]++;
																	}
																} else {
																	if (F4[i] <= 0.09) {
																		if (F1[i] <= 0.66) {
																			if (F3[i] <= 1.85) {
																				Class[2]++;
																			} else {
																				if (F0[i] <= 7.2) {
																					Class[3]++;
																				} else {
																					Class[2]++;
																				}
																			}
																		} else {
																			Class[2]++;
																		}
																	} else {
																		if (F8[i] <= 3.47) {
																			if (F0[i] <= 8.3) {
																				Class[3]++;
																			} else {
																				if (F8[i] <= 3.23) {
																					Class[1]++;
																				} else {
																					Class[2]++;
																				}
																			}
																		} else {
																			Class[2]++;
																		}
																	}
																}
															} else {
																Class[2]++;
															}
														} else {
															if (F6[i] <= 29.5) {
																Class[2]++;
															} else {
																if (F7[i] <= 0.999) {
																	Class[3]++;
																} else {
																	Class[2]++;
																}
															}
														}
													}
												}
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										Class[2]++;
									}
								}
							} else {
								if (F6[i] <= 17.5) {
									Class[4]++;
								} else {
									if (F3[i] <= 1.35) {
										Class[1]++;
									} else {
										if (F9[i] <= 1.08) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									}
								}
							}
						} else {
							if (F8[i] <= 2.915) {
								if (F3[i] <= 2.8) {
									Class[3]++;
								} else {
									Class[1]++;
								}
							} else {
								if (F3[i] <= 2.35) {
									if (F7[i] <= 0.998) {
										if (F6[i] <= 34.5) {
											if (F2[i] <= 0.485) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F4[i] <= 0.174) {
										if (F3[i] <= 2.55) {
											Class[3]++;
										} else {
											if (F10[i] <= 10.0) {
												Class[2]++;
											} else {
												if (F2[i] <= 0.115) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											}
										}
									} else {
										Class[3]++;
									}
								}
							}
						}
					}
				} else {
					if (F5[i] <= 28.5) {
						if (F8[i] <= 3.075) {
							if (F1[i] <= 0.3) {
								Class[4]++;
							} else {
								if (F1[i] <= 0.35) {
									if (F10[i] <= 9.65) {
										Class[3]++;
									} else {
										Class[5]++;
									}
								} else {
									if (F5[i] <= 21.5) {
										if (F7[i] <= 1.001) {
											if (F7[i] <= 0.997) {
												Class[3]++;
											} else {
												if (F10[i] <= 10.25) {
													if (F1[i] <= 0.41) {
														if (F7[i] <= 0.998) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												} else {
													if (F1[i] <= 0.51) {
														Class[3]++;
													} else {
														Class[1]++;
													}
												}
											}
										} else {
											Class[4]++;
										}
									} else {
										Class[4]++;
									}
								}
							}
						} else {
							if (F7[i] <= 0.997) {
								Class[4]++;
							} else {
								if (F4[i] <= 0.14) {
									if (F1[i] <= 0.385) {
										if (F10[i] <= 10.45) {
											if (F9[i] <= 0.615) {
												if (F6[i] <= 51.0) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											} else {
												if (F2[i] <= 0.42) {
													Class[2]++;
												} else {
													if (F2[i] <= 0.565) {
														Class[3]++;
													} else {
														if (F10[i] <= 9.65) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													}
												}
											}
										} else {
											Class[4]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F6[i] <= 16.0) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								}
							}
						}
					} else {
						if (F7[i] <= 0.999) {
							Class[0]++;
						} else {
							Class[2]++;
						}
					}
				}
			} else {
				if (F9[i] <= 0.625) {
					if (F2[i] <= 0.61) {
						if (F6[i] <= 10.5) {
							if (F1[i] <= 0.375) {
								Class[4]++;
							} else {
								if (F8[i] <= 3.245) {
									Class[4]++;
								} else {
									if (F2[i] <= 0.165) {
										Class[3]++;
									} else {
										if (F7[i] <= 0.996) {
											Class[1]++;
										} else {
											Class[3]++;
										}
									}
								}
							}
						} else {
							if (F5[i] <= 6.5) {
								if (F4[i] <= 0.103) {
									if (F1[i] <= 0.52) {
										if (F1[i] <= 0.355) {
											if (F0[i] <= 7.15) {
												Class[1]++;
											} else {
												Class[4]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F9[i] <= 0.595) {
											if (F10[i] <= 10.65) {
												Class[3]++;
											} else {
												if (F3[i] <= 3.6) {
													Class[2]++;
												} else {
													Class[1]++;
												}
											}
										} else {
											if (F0[i] <= 7.65) {
												Class[1]++;
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								if (F1[i] <= 0.535) {
									if (F10[i] <= 11.0) {
										if (F9[i] <= 0.585) {
											if (F4[i] <= 0.069) {
												Class[1]++;
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F6[i] <= 44.5) {
										if (F2[i] <= 0.13) {
											if (F3[i] <= 3.4) {
												if (F8[i] <= 3.57) {
													Class[3]++;
												} else {
													Class[1]++;
												}
											} else {
												Class[1]++;
											}
										} else {
											if (F10[i] <= 11.35) {
												Class[2]++;
											} else {
												Class[4]++;
											}
										}
									} else {
										if (F9[i] <= 0.565) {
											Class[4]++;
										} else {
											if (F5[i] <= 24.0) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									}
								}
							}
						}
					} else {
						if (F5[i] <= 16.0) {
							Class[2]++;
						} else {
							Class[3]++;
						}
					}
				} else {
					if (F2[i] <= 0.505) {
						if (F7[i] <= 0.994) {
							if (F7[i] <= 0.992) {
								if (F6[i] <= 55.0) {
									Class[5]++;
								} else {
									if (F7[i] <= 0.991) {
										if (F6[i] <= 83.5) {
											Class[3]++;
										} else {
											Class[5]++;
										}
									} else {
										Class[3]++;
									}
								}
							} else {
								if (F2[i] <= 0.005) {
									Class[2]++;
								} else {
									if (F3[i] <= 1.55) {
										if (F10[i] <= 12.95) {
											Class[5]++;
										} else {
											Class[4]++;
										}
									} else {
										if (F8[i] <= 3.6) {
											Class[4]++;
										} else {
											Class[2]++;
										}
									}
								}
							}
						} else {
							if (F10[i] <= 11.55) {
								if (F7[i] <= 0.996) {
									if (F8[i] <= 3.285) {
										if (F5[i] <= 17.5) {
											if (F9[i] <= 0.78) {
												Class[5]++;
											} else {
												Class[2]++;
											}
										} else {
											if (F10[i] <= 11.2) {
												Class[3]++;
											} else {
												Class[4]++;
											}
										}
									} else {
										if (F4[i] <= 0.079) {
											Class[3]++;
										} else {
											if (F1[i] <= 0.33) {
												Class[2]++;
											} else {
												if (F10[i] <= 11.0) {
													if (F1[i] <= 0.535) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												} else {
													Class[3]++;
												}
											}
										}
									}
								} else {
									if (F0[i] <= 7.85) {
										if (F7[i] <= 0.997) {
											if (F6[i] <= 33.0) {
												Class[1]++;
											} else {
												if (F2[i] <= 0.405) {
													Class[2]++;
												} else {
													if (F0[i] <= 7.15) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											}
										} else {
											if (F2[i] <= 0.095) {
												Class[1]++;
											} else {
												Class[4]++;
											}
										}
									} else {
										if (F4[i] <= 0.094) {
											if (F8[i] <= 3.09) {
												if (F2[i] <= 0.365) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												if (F4[i] <= 0.058) {
													if (F8[i] <= 3.24) {
														Class[4]++;
													} else {
														Class[2]++;
													}
												} else {
													if (F9[i] <= 0.715) {
														if (F5[i] <= 22.5) {
															Class[4]++;
														} else {
															Class[3]++;
														}
													} else {
														if (F7[i] <= 0.996) {
															Class[4]++;
														} else {
															if (F4[i] <= 0.091) {
																if (F7[i] <= 0.996) {
																	if (F8[i] <= 3.37) {
																		Class[3]++;
																	} else {
																		Class[4]++;
																	}
																} else {
																	if (F9[i] <= 0.98) {
																		Class[3]++;
																	} else {
																		if (F9[i] <= 1.03) {
																			Class[4]++;
																		} else {
																			Class[3]++;
																		}
																	}
																}
															} else {
																Class[4]++;
															}
														}
													}
												}
											}
										} else {
											if (F9[i] <= 0.75) {
												if (F2[i] <= 0.405) {
													Class[2]++;
												} else {
													if (F10[i] <= 11.0) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F7[i] <= 0.998) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											}
										}
									}
								}
							} else {
								if (F3[i] <= 2.15) {
									if (F4[i] <= 0.07) {
										if (F3[i] <= 1.85) {
											Class[5]++;
										} else {
											if (F2[i] <= 0.405) {
												Class[4]++;
											} else {
												Class[3]++;
											}
										}
									} else {
										Class[3]++;
									}
								} else {
									if (F10[i] <= 13.4) {
										if (F1[i] <= 0.235) {
											Class[3]++;
										} else {
											if (F8[i] <= 3.39) {
												if (F8[i] <= 3.225) {
													if (F7[i] <= 0.999) {
														Class[3]++;
													} else {
														Class[4]++;
													}
												} else {
													if (F9[i] <= 0.7) {
														if (F9[i] <= 0.665) {
															Class[4]++;
														} else {
															Class[3]++;
														}
													} else {
														Class[4]++;
													}
												}
											} else {
												if (F3[i] <= 2.75) {
													if (F6[i] <= 26.5) {
														Class[4]++;
													} else {
														if (F9[i] <= 0.68) {
															Class[3]++;
														} else {
															if (F0[i] <= 8.35) {
																Class[5]++;
															} else {
																Class[2]++;
															}
														}
													}
												} else {
													Class[3]++;
												}
											}
										}
									} else {
										Class[3]++;
									}
								}
							}
						}
					} else {
						if (F7[i] <= 1.001) {
							if (F1[i] <= 0.605) {
								if (F8[i] <= 3.37) {
									if (F9[i] <= 0.64) {
										Class[5]++;
									} else {
										if (F10[i] <= 11.55) {
											if (F10[i] <= 10.85) {
												if (F7[i] <= 0.998) {
													Class[2]++;
												} else {
													Class[4]++;
												}
											} else {
												if (F3[i] <= 2.9) {
													if (F2[i] <= 0.7) {
														Class[3]++;
													} else {
														Class[4]++;
													}
												} else {
													Class[2]++;
												}
											}
										} else {
											if (F8[i] <= 2.995) {
												Class[2]++;
											} else {
												if (F0[i] <= 9.5) {
													if (F10[i] <= 11.75) {
														Class[5]++;
													} else {
														Class[4]++;
													}
												} else {
													Class[4]++;
												}
											}
										}
									}
								} else {
									if (F0[i] <= 7.9) {
										Class[3]++;
									} else {
										Class[2]++;
									}
								}
							} else {
								Class[5]++;
							}
						} else {
							Class[3]++;
						}
					}
				}
			}
		}

		if (F1[i] <= 0.363) {
			if (F6[i] <= 75.5) {
				if (F10[i] <= 10.45) {
					if (F10[i] <= 9.75) {
						if (F9[i] <= 0.585) {
							Class[2]++;
						} else {
							if (F5[i] <= 10.5) {
								if (F4[i] <= 0.076) {
									if (F1[i] <= 0.235) {
										Class[4]++;
									} else {
										Class[3]++;
									}
								} else {
									if (F0[i] <= 10.95) {
										Class[2]++;
									} else {
										Class[3]++;
									}
								}
							} else {
								if (F7[i] <= 0.996) {
									Class[2]++;
								} else {
									Class[3]++;
								}
							}
						}
					} else {
						if (F8[i] <= 3.345) {
							if (F2[i] <= 0.66) {
								if (F5[i] <= 3.5) {
									Class[5]++;
								} else {
									if (F3[i] <= 1.85) {
										if (F0[i] <= 11.45) {
											Class[3]++;
										} else {
											Class[4]++;
										}
									} else {
										Class[3]++;
									}
								}
							} else {
								Class[5]++;
							}
						} else {
							if (F6[i] <= 54.5) {
								if (F5[i] <= 10.0) {
									Class[2]++;
								} else {
									if (F9[i] <= 0.62) {
										Class[1]++;
									} else {
										if (F8[i] <= 3.39) {
											Class[3]++;
										} else {
											Class[4]++;
										}
									}
								}
							} else {
								if (F9[i] <= 0.82) {
									Class[3]++;
								} else {
									if (F0[i] <= 9.25) {
										Class[4]++;
									} else {
										Class[3]++;
									}
								}
							}
						}
					}
				} else {
					if (F9[i] <= 0.545) {
						Class[3]++;
					} else {
						if (F6[i] <= 67.5) {
							if (F8[i] <= 3.335) {
								if (F1[i] <= 0.275) {
									if (F4[i] <= 0.082) {
										if (F3[i] <= 3.55) {
											if (F4[i] <= 0.071) {
												if (F10[i] <= 11.7) {
													Class[3]++;
												} else {
													Class[4]++;
												}
											} else {
												Class[4]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F4[i] <= 0.09) {
											Class[2]++;
										} else {
											if (F7[i] <= 0.997) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										}
									}
								} else {
									if (F1[i] <= 0.345) {
										if (F2[i] <= 0.485) {
											if (F7[i] <= 0.995) {
												Class[4]++;
											} else {
												if (F4[i] <= 0.077) {
													Class[4]++;
												} else {
													Class[3]++;
												}
											}
										} else {
											if (F8[i] <= 3.23) {
												if (F1[i] <= 0.315) {
													if (F7[i] <= 0.997) {
														if (F4[i] <= 0.07) {
															Class[4]++;
														} else {
															Class[5]++;
														}
													} else {
														Class[4]++;
													}
												} else {
													Class[2]++;
												}
											} else {
												if (F3[i] <= 2.65) {
													if (F4[i] <= 0.061) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												} else {
													Class[4]++;
												}
											}
										}
									} else {
										if (F8[i] <= 3.265) {
											if (F3[i] <= 1.65) {
												Class[4]++;
											} else {
												if (F9[i] <= 0.745) {
													Class[5]++;
												} else {
													Class[2]++;
												}
											}
										} else {
											Class[3]++;
										}
									}
								}
							} else {
								if (F7[i] <= 0.998) {
									if (F3[i] <= 1.75) {
										Class[3]++;
									} else {
										if (F5[i] <= 6.0) {
											Class[4]++;
										} else {
											if (F8[i] <= 3.395) {
												Class[3]++;
											} else {
												if (F5[i] <= 25.5) {
													if (F0[i] <= 7.15) {
														if (F10[i] <= 10.8) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													} else {
														Class[4]++;
													}
												} else {
													Class[3]++;
												}
											}
										}
									}
								} else {
									Class[2]++;
								}
							}
						} else {
							if (F5[i] <= 35.0) {
								Class[2]++;
							} else {
								Class[5]++;
							}
						}
					}
				}
			} else {
				if (F1[i] <= 0.305) {
					if (F3[i] <= 1.75) {
						if (F4[i] <= 0.047) {
							Class[2]++;
						} else {
							Class[3]++;
						}
					} else {
						Class[4]++;
					}
				} else {
					Class[2]++;
				}
			}
		} else {
			if (F9[i] <= 0.555) {
				if (F2[i] <= 0.125) {
					if (F8[i] <= 3.275) {
						Class[2]++;
					} else {
						if (F7[i] <= 0.996) {
							if (F10[i] <= 12.1) {
								if (F5[i] <= 29.0) {
									if (F4[i] <= 0.074) {
										if (F7[i] <= 0.994) {
											Class[2]++;
										} else {
											if (F10[i] <= 10.2) {
												if (F4[i] <= 0.061) {
													Class[2]++;
												} else {
													if (F3[i] <= 1.4) {
														Class[4]++;
													} else {
														Class[1]++;
													}
												}
											} else {
												Class[1]++;
											}
										}
									} else {
										if (F2[i] <= 0.005) {
											if (F10[i] <= 11.05) {
												if (F8[i] <= 3.365) {
													Class[0]++;
												} else {
													if (F10[i] <= 10.85) {
														Class[2]++;
													} else {
														Class[0]++;
													}
												}
											} else {
												if (F5[i] <= 7.5) {
													Class[3]++;
												} else {
													if (F4[i] <= 0.087) {
														Class[1]++;
													} else {
														Class[4]++;
													}
												}
											}
										} else {
											if (F1[i] <= 0.755) {
												if (F5[i] <= 21.5) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												Class[1]++;
											}
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F9[i] <= 0.545) {
								if (F2[i] <= 0.115) {
									if (F10[i] <= 9.55) {
										Class[2]++;
									} else {
										if (F0[i] <= 7.35) {
											if (F1[i] <= 0.805) {
												if (F7[i] <= 0.997) {
													if (F8[i] <= 3.505) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												} else {
													Class[2]++;
												}
											} else {
												if (F2[i] <= 0.025) {
													Class[2]++;
												} else {
													Class[0]++;
												}
											}
										} else {
											if (F7[i] <= 0.996) {
												Class[3]++;
											} else {
												if (F8[i] <= 3.35) {
													Class[1]++;
												} else {
													if (F4[i] <= 0.082) {
														if (F2[i] <= 0.005) {
															Class[1]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												}
											}
										}
									}
								} else {
									Class[3]++;
								}
							} else {
								if (F3[i] <= 2.2) {
									Class[0]++;
								} else {
									if (F8[i] <= 3.305) {
										Class[2]++;
									} else {
										if (F0[i] <= 7.5) {
											if (F4[i] <= 0.09) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[3]++;
										}
									}
								}
							}
						}
					}
				} else {
					if (F5[i] <= 4.5) {
						Class[3]++;
					} else {
						if (F10[i] <= 11.75) {
							if (F5[i] <= 12.5) {
								if (F1[i] <= 0.593) {
									if (F10[i] <= 9.125) {
										Class[1]++;
									} else {
										if (F3[i] <= 3.2) {
											if (F8[i] <= 3.335) {
												if (F1[i] <= 0.58) {
													if (F5[i] <= 9.5) {
														Class[2]++;
													} else {
														if (F4[i] <= 0.07) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													}
												} else {
													Class[3]++;
												}
											} else {
												if (F9[i] <= 0.505) {
													Class[2]++;
												} else {
													Class[1]++;
												}
											}
										} else {
											Class[3]++;
										}
									}
								} else {
									Class[2]++;
								}
							} else {
								if (F5[i] <= 16.5) {
									if (F4[i] <= 0.082) {
										if (F2[i] <= 0.21) {
											if (F10[i] <= 9.75) {
												Class[3]++;
											} else {
												Class[2]++;
											}
										} else {
											Class[3]++;
										}
									} else {
										if (F6[i] <= 34.0) {
											if (F10[i] <= 9.8) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											Class[2]++;
										}
									}
								} else {
									if (F4[i] <= 0.091) {
										if (F9[i] <= 0.535) {
											if (F1[i] <= 0.55) {
												if (F9[i] <= 0.47) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												Class[2]++;
											}
										} else {
											if (F0[i] <= 8.1) {
												if (F5[i] <= 25.0) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											} else {
												Class[2]++;
											}
										}
									} else {
										if (F8[i] <= 3.15) {
											Class[2]++;
										} else {
											if (F3[i] <= 2.9) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									}
								}
							}
						} else {
							if (F6[i] <= 12.5) {
								Class[3]++;
							} else {
								Class[4]++;
							}
						}
					}
				}
			} else {
				if (F8[i] <= 3.265) {
					if (F10[i] <= 11.45) {
						if (F8[i] <= 2.805) {
							Class[1]++;
						} else {
							if (F7[i] <= 1.0) {
								if (F6[i] <= 36.0) {
									if (F4[i] <= 0.081) {
										if (F8[i] <= 3.075) {
											if (F3[i] <= 1.7) {
												Class[1]++;
											} else {
												Class[2]++;
											}
										} else {
											if (F1[i] <= 0.455) {
												if (F9[i] <= 0.75) {
													if (F8[i] <= 3.255) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												} else {
													Class[4]++;
												}
											} else {
												if (F8[i] <= 3.13) {
													if (F3[i] <= 2.45) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												} else {
													Class[2]++;
												}
											}
										}
									} else {
										if (F1[i] <= 0.49) {
											if (F7[i] <= 1.0) {
												Class[3]++;
											} else {
												Class[4]++;
											}
										} else {
											if (F7[i] <= 1.0) {
												if (F4[i] <= 0.085) {
													Class[3]++;
												} else {
													if (F7[i] <= 0.996) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												}
											} else {
												Class[3]++;
											}
										}
									}
								} else {
									if (F8[i] <= 2.965) {
										if (F3[i] <= 1.95) {
											Class[3]++;
										} else {
											if (F0[i] <= 8.9) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									} else {
										if (F10[i] <= 9.75) {
											if (F10[i] <= 9.15) {
												Class[3]++;
											} else {
												if (F9[i] <= 0.585) {
													if (F7[i] <= 0.997) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												} else {
													Class[2]++;
												}
											}
										} else {
											if (F9[i] <= 0.72) {
												if (F1[i] <= 0.545) {
													if (F4[i] <= 0.075) {
														Class[3]++;
													} else {
														if (F5[i] <= 31.5) {
															if (F9[i] <= 0.685) {
																Class[2]++;
															} else {
																if (F1[i] <= 0.415) {
																	Class[4]++;
																} else {
																	Class[2]++;
																}
															}
														} else {
															if (F2[i] <= 0.265) {
																Class[3]++;
															} else {
																Class[2]++;
															}
														}
													}
												} else {
													if (F9[i] <= 0.645) {
														if (F1[i] <= 0.827) {
															Class[3]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[2]++;
													}
												}
											} else {
												if (F7[i] <= 1.0) {
													if (F8[i] <= 3.095) {
														if (F1[i] <= 0.49) {
															Class[4]++;
														} else {
															Class[2]++;
														}
													} else {
														Class[3]++;
													}
												} else {
													Class[2]++;
												}
											}
										}
									}
								}
							} else {
								Class[3]++;
							}
						}
					} else {
						if (F9[i] <= 0.615) {
							if (F5[i] <= 8.0) {
								if (F8[i] <= 3.18) {
									Class[4]++;
								} else {
									Class[3]++;
								}
							} else {
								Class[3]++;
							}
						} else {
							if (F2[i] <= 0.565) {
								if (F0[i] <= 9.5) {
									Class[5]++;
								} else {
									if (F7[i] <= 0.996) {
										Class[4]++;
									} else {
										Class[3]++;
									}
								}
							} else {
								if (F1[i] <= 0.583) {
									if (F6[i] <= 76.5) {
										Class[4]++;
									} else {
										Class[2]++;
									}
								} else {
									Class[3]++;
								}
							}
						}
					}
				} else {
					if (F10[i] <= 10.55) {
						if (F10[i] <= 10.45) {
							if (F5[i] <= 26.5) {
								if (F8[i] <= 3.385) {
									if (F0[i] <= 6.75) {
										if (F2[i] <= 0.32) {
											Class[2]++;
										} else {
											Class[3]++;
										}
									} else {
										if (F1[i] <= 0.945) {
											if (F2[i] <= 0.475) {
												if (F4[i] <= 0.072) {
													Class[2]++;
												} else {
													if (F10[i] <= 9.75) {
														if (F6[i] <= 95.0) {
															if (F4[i] <= 0.087) {
																if (F2[i] <= 0.195) {
																	if (F5[i] <= 7.5) {
																		Class[3]++;
																	} else {
																		if (F3[i] <= 2.05) {
																			if (F1[i] <= 0.51) {
																				Class[2]++;
																			} else {
																				Class[4]++;
																			}
																		} else {
																			Class[3]++;
																		}
																	}
																} else {
																	if (F7[i] <= 0.999) {
																		Class[2]++;
																	} else {
																		Class[3]++;
																	}
																}
															} else {
																if (F6[i] <= 26.5) {
																	if (F4[i] <= 0.109) {
																		if (F9[i] <= 0.685) {
																			if (F6[i] <= 17.0) {
																				Class[3]++;
																			} else {
																				Class[2]++;
																			}
																		} else {
																			Class[2]++;
																		}
																	} else {
																		Class[3]++;
																	}
																} else {
																	Class[3]++;
																}
															}
														} else {
															Class[2]++;
														}
													} else {
														if (F4[i] <= 0.131) {
															if (F8[i] <= 3.295) {
																if (F6[i] <= 27.5) {
																	Class[2]++;
																} else {
																	if (F6[i] <= 62.5) {
																		Class[3]++;
																	} else {
																		if (F1[i] <= 0.645) {
																			Class[3]++;
																		} else {
																			Class[2]++;
																		}
																	}
																}
															} else {
																if (F1[i] <= 0.602) {
																	Class[3]++;
																} else {
																	if (F2[i] <= 0.3) {
																		Class[3]++;
																	} else {
																		Class[1]++;
																	}
																}
															}
														} else {
															Class[4]++;
														}
													}
												}
											} else {
												Class[2]++;
											}
										} else {
											Class[1]++;
										}
									}
								} else {
									if (F7[i] <= 0.996) {
										if (F2[i] <= 0.195) {
											if (F1[i] <= 0.41) {
												if (F1[i] <= 0.39) {
													Class[2]++;
												} else {
													Class[4]++;
												}
											} else {
												if (F3[i] <= 1.7) {
													Class[2]++;
												} else {
													Class[3]++;
												}
											}
										} else {
											Class[2]++;
										}
									} else {
										if (F4[i] <= 0.067) {
											if (F0[i] <= 6.8) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										} else {
											if (F10[i] <= 9.35) {
												if (F9[i] <= 0.625) {
													if (F1[i] <= 0.635) {
														Class[3]++;
													} else {
														Class[2]++;
													}
												} else {
													Class[3]++;
												}
											} else {
												if (F1[i] <= 0.685) {
													if (F1[i] <= 0.675) {
														if (F0[i] <= 6.3) {
															Class[3]++;
														} else {
															if (F3[i] <= 2.15) {
																if (F7[i] <= 0.997) {
																	if (F3[i] <= 1.6) {
																		Class[2]++;
																	} else {
																		Class[3]++;
																	}
																} else {
																	Class[2]++;
																}
															} else {
																if (F10[i] <= 10.0) {
																	if (F8[i] <= 3.44) {
																		Class[3]++;
																	} else {
																		if (F1[i] <= 0.615) {
																			Class[2]++;
																		} else {
																			if (F1[i] <= 0.64) {
																				Class[3]++;
																			} else {
																				Class[2]++;
																			}
																		}
																	}
																} else {
																	Class[4]++;
																}
															}
														}
													} else {
														Class[3]++;
													}
												} else {
													Class[2]++;
												}
											}
										}
									}
								}
							} else {
								if (F8[i] <= 3.305) {
									if (F5[i] <= 30.5) {
										Class[3]++;
									} else {
										Class[4]++;
									}
								} else {
									if (F3[i] <= 1.75) {
										if (F6[i] <= 63.5) {
											Class[0]++;
										} else {
											Class[3]++;
										}
									} else {
										if (F1[i] <= 0.385) {
											Class[3]++;
										} else {
											if (F1[i] <= 0.688) {
												if (F9[i] <= 0.595) {
													if (F9[i] <= 0.585) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												} else {
													Class[2]++;
												}
											} else {
												if (F4[i] <= 0.09) {
													Class[3]++;
												} else {
													Class[2]++;
												}
											}
										}
									}
								}
							}
						} else {
							if (F9[i] <= 0.875) {
								if (F2[i] <= 0.34) {
									if (F9[i] <= 0.75) {
										if (F7[i] <= 0.997) {
											Class[3]++;
										} else {
											Class[2]++;
										}
									} else {
										Class[4]++;
									}
								} else {
									if (F1[i] <= 0.51) {
										Class[2]++;
									} else {
										Class[3]++;
									}
								}
							} else {
								Class[4]++;
							}
						}
					} else {
						if (F6[i] <= 88.5) {
							if (F9[i] <= 0.575) {
								if (F5[i] <= 20.5) {
									if (F7[i] <= 0.995) {
										if (F2[i] <= 0.05) {
											Class[2]++;
										} else {
											if (F8[i] <= 3.53) {
												Class[2]++;
											} else {
												Class[3]++;
											}
										}
									} else {
										if (F2[i] <= 0.155) {
											Class[1]++;
										} else {
											if (F8[i] <= 3.325) {
												Class[1]++;
											} else {
												Class[2]++;
											}
										}
									}
								} else {
									if (F0[i] <= 6.65) {
										Class[4]++;
									} else {
										Class[3]++;
									}
								}
							} else {
								if (F9[i] <= 0.685) {
									if (F1[i] <= 1.062) {
										if (F2[i] <= 0.005) {
											if (F3[i] <= 2.175) {
												if (F3[i] <= 1.75) {
													Class[3]++;
												} else {
													if (F5[i] <= 6.5) {
														Class[2]++;
													} else {
														Class[3]++;
													}
												}
											} else {
												if (F8[i] <= 3.665) {
													if (F5[i] <= 11.0) {
														if (F4[i] <= 0.067) {
															Class[2]++;
														} else {
															Class[1]++;
														}
													} else {
														Class[3]++;
													}
												} else {
													Class[4]++;
												}
											}
										} else {
											if (F2[i] <= 0.295) {
												if (F10[i] <= 11.65) {
													if (F5[i] <= 6.5) {
														if (F10[i] <= 10.75) {
															Class[2]++;
														} else {
															Class[3]++;
														}
													} else {
														Class[3]++;
													}
												} else {
													if (F10[i] <= 12.25) {
														Class[4]++;
													} else {
														Class[3]++;
													}
												}
											} else {
												if (F3[i] <= 2.55) {
													Class[3]++;
												} else {
													if (F1[i] <= 0.475) {
														Class[3]++;
													} else {
														if (F9[i] <= 0.645) {
															Class[2]++;
														} else {
															Class[4]++;
														}
													}
												}
											}
										}
									} else {
										Class[1]++;
									}
								} else {
									if (F9[i] <= 0.735) {
										if (F5[i] <= 8.0) {
											if (F10[i] <= 11.35) {
												Class[3]++;
											} else {
												Class[5]++;
											}
										} else {
											if (F1[i] <= 0.47) {
												if (F10[i] <= 12.35) {
													Class[3]++;
												} else {
													Class[4]++;
												}
											} else {
												if (F4[i] <= 0.069) {
													Class[3]++;
												} else {
													if (F4[i] <= 0.083) {
														Class[4]++;
													} else {
														if (F10[i] <= 11.4) {
															Class[2]++;
														} else {
															Class[4]++;
														}
													}
												}
											}
										}
									} else {
										if (F4[i] <= 0.06) {
											if (F2[i] <= 0.37) {
												if (F9[i] <= 0.83) {
													Class[5]++;
												} else {
													Class[4]++;
												}
											} else {
												Class[4]++;
											}
										} else {
											if (F3[i] <= 3.5) {
												if (F10[i] <= 12.45) {
													Class[3]++;
												} else {
													if (F6[i] <= 23.0) {
														Class[3]++;
													} else {
														if (F8[i] <= 3.365) {
															Class[4]++;
														} else {
															Class[5]++;
														}
													}
												}
											} else {
												Class[4]++;
											}
										}
									}
								}
							}
						} else {
							if (F8[i] <= 3.435) {
								Class[2]++;
							} else {
								if (F0[i] <= 5.5) {
									Class[4]++;
								} else {
									Class[2]++;
								}
							}
						}
					}
				}
			}
		}
	}
