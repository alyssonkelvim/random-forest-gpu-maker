#include "hip/hip_runtime.h"
__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, float *F6, float *F7, float *F8, float *F9, float *F10, float *F11, float *F12, float *F13, float *F14, float *F15, float *F16, float *F17, float *F18, float *F19, float *F20, float *F21, float *F22, float *F23, float *F24, float *F25, float *F26, float *F27, float *F28, float *F29, float *F30, float *F31, float *F32, float *F33, float *F34, float *F35, float *F36, float *F37, float *F38, float *F39, float *F40, float *F41, float *F42, float *F43, float *F44, float *F45, int *P, const int N)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[2]; 
	Class[0] = 0;
	Class[1] = 0;
	if (i < N) {
		if (F29[i] <= -48.777) {
			if (F2[i] <= 0.089) {
				Class[1]++;
			} else {
				if (F13[i] <= -18.409) {
					if (F13[i] <= -20.773) {
						Class[1]++;
					} else {
						if (F38[i] <= 0.094) {
							if (F0[i] <= 37.66) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						} else {
							if (F39[i] <= 9.594) {
								Class[1]++;
							} else {
								Class[0]++;
							}
						}
					}
				} else {
					if (F45[i] <= -58.259) {
						Class[0]++;
					} else {
						Class[1]++;
					}
				}
			}
		} else {
			if (F30[i] <= 45.19) {
				Class[1]++;
			} else {
				if (F30[i] <= 45.3) {
					Class[0]++;
				} else {
					if (F29[i] <= -48.624) {
						if (F2[i] <= 1.087) {
							Class[1]++;
						} else {
							Class[0]++;
						}
					} else {
						Class[1]++;
					}
				}
			}
		}

		if (F20[i] <= 9.495) {
			if (F20[i] <= 9.458) {
				if (F45[i] <= -59.612) {
					if (F23[i] <= 9.44) {
						Class[1]++;
					} else {
						Class[0]++;
					}
				} else {
					Class[1]++;
				}
			} else {
				if (F35[i] <= 30.781) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			}
		} else {
			if (F28[i] <= 2.456) {
				if (F40[i] <= -1.336) {
					if (F46[i] <= -37.184) {
						if (F8[i] <= 0.131) {
							if (F0[i] <= 37.625) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						} else {
							if (F25[i] <= 0.003) {
								if (F22[i] <= 0.098) {
									if (F0[i] <= 36.345) {
										Class[0]++;
									} else {
										Class[1]++;
									}
								} else {
									if (F13[i] <= -16.157) {
										Class[1]++;
									} else {
										Class[0]++;
									}
								}
							} else {
								Class[0]++;
							}
						}
					} else {
						if (F3[i] <= 7.632) {
							if (F4[i] <= 5.743) {
								if (F45[i] <= -60.662) {
									if (F47[i] <= -58.866) {
										Class[1]++;
									} else {
										if (F43[i] <= -0.015) {
											if (F28[i] <= 0.974) {
												Class[0]++;
											} else {
												Class[1]++;
											}
										} else {
											Class[0]++;
										}
									}
								} else {
									Class[1]++;
								}
							} else {
								if (F11[i] <= 7.422) {
									Class[0]++;
								} else {
									Class[1]++;
								}
							}
						} else {
							if (F3[i] <= 7.673) {
								if (F2[i] <= 2.29) {
									Class[0]++;
								} else {
									Class[1]++;
								}
							} else {
								if (F13[i] <= -20.262) {
									if (F38[i] <= 0.075) {
										Class[1]++;
									} else {
										Class[0]++;
									}
								} else {
									if (F38[i] <= 0.095) {
										Class[0]++;
									} else {
										if (F7[i] <= 6.158) {
											Class[0]++;
										} else {
											Class[1]++;
										}
									}
								}
							}
						}
					}
				} else {
					Class[1]++;
				}
			} else {
				if (F35[i] <= 30.594) {
					Class[0]++;
				} else {
					Class[1]++;
				}
			}
		}

		if (F23[i] <= 9.539) {
			if (F0[i] <= 37.075) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F35[i] <= 30.781) {
				if (F35[i] <= 30.719) {
					Class[0]++;
				} else {
					if (F2[i] <= 0.102) {
						Class[1]++;
					} else {
						if (F6[i] <= 7.406) {
							if (F0[i] <= 37.145) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						} else {
							if (F0[i] <= 37.66) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						}
					}
				}
			} else {
				Class[1]++;
			}
		}

		if (F45[i] <= -59.158) {
			if (F47[i] <= -55.196) {
				if (F6[i] <= 7.778) {
					if (F13[i] <= -18.793) {
						if (F12[i] <= -67.203) {
							if (F41[i] <= 0.243) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						} else {
							Class[1]++;
						}
					} else {
						if (F0[i] <= 38.205) {
							Class[0]++;
						} else {
							Class[1]++;
						}
					}
				} else {
					if (F0[i] <= 37.655) {
						Class[0]++;
					} else {
						Class[1]++;
					}
				}
			} else {
				Class[1]++;
			}
		} else {
			if (F0[i] <= 27.975) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		}

		if (F23[i] <= 9.539) {
			if (F23[i] <= 9.479) {
				if (F12[i] <= -68.786) {
					if (F5[i] <= -0.191) {
						Class[1]++;
					} else {
						Class[0]++;
					}
				} else {
					if (F1[i] <= 30.281) {
						Class[0]++;
					} else {
						Class[1]++;
					}
				}
			} else {
				if (F35[i] <= 30.781) {
					if (F4[i] <= 4.936) {
						Class[0]++;
					} else {
						Class[1]++;
					}
				} else {
					Class[1]++;
				}
			}
		} else {
			if (F0[i] <= 37.65) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		}

		if (F20[i] <= 9.495) {
			if (F18[i] <= 32.219) {
				Class[0]++;
			} else {
				Class[1]++;
			}
		} else {
			if (F35[i] <= 30.781) {
				if (F6[i] <= 7.42) {
					if (F12[i] <= -68.198) {
						Class[0]++;
					} else {
						if (F11[i] <= 6.895) {
							Class[0]++;
						} else {
							if (F19[i] <= 0.484) {
								Class[1]++;
							} else {
								Class[0]++;
							}
						}
					}
				} else {
					if (F18[i] <= 32.219) {
						if (F40[i] <= -1.752) {
							if (F0[i] <= 37.66) {
								Class[0]++;
							} else {
								Class[1]++;
							}
						} else {
							if (F23[i] <= 9.801) {
								if (F0[i] <= 37.705) {
									Class[0]++;
								} else {
									Class[1]++;
								}
							} else {
								if (F19[i] <= 0.144) {
									Class[1]++;
								} else {
									Class[0]++;
								}
							}
						}
					} else {
						Class[1]++;
					}
				}
			} else {
				Class[1]++;
			}
		}
	}
}