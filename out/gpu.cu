#include "hip/hip_runtime.h"
__global__ void RF_with_EQ(float *A, float *B, float *C, float *D, int *P,const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int Class[4];
    Class[0] = 0;
    Class[1] = 0;
    Class[2] = 0;
    Class[3] = 0;
    if (i < N) {
        bool t1,t2,t3,t4,a1,a2,a3,a4,b1,b2,b3,c1,c2,c3;
        t1= A[i] > 3.0;
        t2 = B[i] > 10.0;
        t3 = C[i] > 5.0;
        t4 = D[i] > 1.0;Medição via Clock
        a1 = B[i] > 7.0;
        a2 = A[i] > 10.0;
        a3 = D[i] > 5.0;
        a4 = C[i] > 1.0;
        b1 = A[i] > 3.0;
        b2 = C[i] > 10.0;
        b3 = D[i] > 10.0;
        c1 = D[i] > 7.0;
        c2 = D[i] > 5.0;
        c3 = C[i] > 1.0;
        Class[0] = (t1&&t2)+(!t1&&!t3&&t4)+(!a1&&a3)+(b1&&b2)+(!b1&&b3)+c1;
        Class[1] =(!t1&&t3)+(a1&&a2)+(!a1&&!a3&&!a4)+(!c1&&!c2);
        Class[2] = (t1&&!t2)+(!a1&&!a3&&a4)+(b1&&!b2)+(!b1&&!b3)+(!c1&&c2&&!c3);
        Class[3] =(!t1&&!t3&&!t4)+(a1&&!a2)+(!c1&&c2&&c3);
        int p0,p1,Q0,Q1;
        p0 = (Class[1] > Class[0])?1:0;
        Q0 = (Class[1] > Class[0])?Class[1]:Class[0];
        p1= (Class[2] > Class[3])?2:3;
        Q1 = (Class[2] > Class[3])?Class[2]:Class[3];
        P[i] = (Q1 > Q0)? p1:p0;
    }    

}