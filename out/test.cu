#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}

FILE *inFile;
FILE *outFile;

void openInFile(){
    inFile = fopen("in/dataset.csv","r");
    if (inFile == NULL){
        printf("Erro ao tentar abrir o arquivo!");
    }
}

void closeInFile(){
    fclose(inFile);
}

void readInFile(float *ip, int index){ 
    char c;
    int i = 0, j = 0, readIndex = 0;
    char line[50];
    outFile = fopen("out/out_rf_with_if.csv","a");
    printf("Lendo e exibindo os dados do arquivo \n\n");
    c = fgetc(inFile);
    while (c != EOF){
        if(c == ',' || c == '\n'){
            if(index == readIndex){
                ip[i] = atof(line);        
                i++;
            }
            line = "00";
            j = 0;
            readIndex++;
        }else{
            line[j] = c;
            j++;
        }
        
        if(c == '\n'){
            readIndex = 0;
        }
        c = fgetc(inFile);
    }
    fclose(outFile);
}

void writeOutFile(int value){
    fprintf(outFile, "%d\n", value);
}

int main(int argc, char ** argv) {
    openInFile();
    openOutFile();
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 25;
    printf("Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4,  * h_5,  * h_6,  * h_7,  * h_8,  * h_9,  * h_10;
    int * h_P, * hostRef;
    	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
	h_5 = (float * ) malloc(nBytes);
	h_6 = (float * ) malloc(nBytes);
	h_7 = (float * ) malloc(nBytes);
	h_8 = (float * ) malloc(nBytes);
	h_9 = (float * ) malloc(nBytes);
	h_10 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

    	readInFile(h_0, nElem);
	readInFile(h_1, nElem);
	readInFile(h_2, nElem);
	readInFile(h_3, nElem);
	readInFile(h_4, nElem);
	readInFile(h_5, nElem);
	readInFile(h_6, nElem);
	readInFile(h_7, nElem);
	readInFile(h_8, nElem);
	readInFile(h_9, nElem);
	readInFile(h_10, nElem);

    closeInFile();
    
    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4, * d_5, * d_6, * d_7, * d_8, * d_9, * d_10;
    int * d_P;
        CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((float ** ) & d_5, nBytes));
    CHECK(hipMalloc((float ** ) & d_6, nBytes));
    CHECK(hipMalloc((float ** ) & d_7, nBytes));
    CHECK(hipMalloc((float ** ) & d_8, nBytes));
    CHECK(hipMalloc((float ** ) & d_9, nBytes));
    CHECK(hipMalloc((float ** ) & d_10, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));


    // transfer data from host to device
        CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_6, h_6, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_7, h_7, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_8, h_8, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_9, h_9, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_10, h_10, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_IF << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_P, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("RF with IF - execution time = %.6fms\n", elapsed_time);

    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_EQ << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_P, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("RF with EQ - execution time = %.6fms\n", elapsed_time);

    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
        free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(h_5);
    free(h_6);
    free(h_7);
    free(h_8);
    free(h_9);
    free(h_10);
    free(hostRef);

    return (0);
 }