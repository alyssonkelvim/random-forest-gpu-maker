#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}int main(int argc, char ** argv) {
     
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 25;
    printf("Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4,  * h_5,  * h_6,  * h_7,  * h_8,  * h_9,  * h_10;
    int * h_P, * hostRef;
    	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
	h_5 = (float * ) malloc(nBytes);
	h_6 = (float * ) malloc(nBytes);
	h_7 = (float * ) malloc(nBytes);
	h_8 = (float * ) malloc(nBytes);
	h_9 = (float * ) malloc(nBytes);
	h_10 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

    	initialData(h_0, nElem);
	initialData(h_1, nElem);
	initialData(h_2, nElem);
	initialData(h_3, nElem);
	initialData(h_4, nElem);
	initialData(h_5, nElem);
	initialData(h_6, nElem);
	initialData(h_7, nElem);
	initialData(h_8, nElem);
	initialData(h_9, nElem);
	initialData(h_10, nElem);

    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4, * d_5, * d_6, * d_7, * d_8, * d_9, * d_10;
    int * d_P;
        CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((float ** ) & d_5, nBytes));
    CHECK(hipMalloc((float ** ) & d_6, nBytes));
    CHECK(hipMalloc((float ** ) & d_7, nBytes));
    CHECK(hipMalloc((float ** ) & d_8, nBytes));
    CHECK(hipMalloc((float ** ) & d_9, nBytes));
    CHECK(hipMalloc((float ** ) & d_10, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));

    // transfer data from host to device
        CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_6, h_6, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_7, h_7, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_8, h_8, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_9, h_9, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_10, h_10, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_IF << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_P, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("RF with IF - execution time = %.6fms\n", elapsed_time);

    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_EQ << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_P, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("RF with EQ - execution time = %.6fms\n", elapsed_time);

    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
        free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(h_5);
    free(h_6);
    free(h_7);
    free(h_8);
    free(h_9);
    free(h_10);
    free(hostRef);

    return (0);
 }