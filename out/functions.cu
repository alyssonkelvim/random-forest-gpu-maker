#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}